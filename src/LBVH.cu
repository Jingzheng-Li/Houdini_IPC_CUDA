#include "hip/hip_runtime.h"

#include <thrust/device_ptr.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <fstream>
#include <iostream>

#include "LBVH.cuh"


// specific idea
// cloth with boundary 0 human body with boundary 2
// CCD and DCD detect at least one vertex with boundary 0
// which means only do cloth self-collision & cloth-body collision


template <class F>
__device__ __host__ inline F __m_min(F a, F b) {
    return a > b ? b : a;
}

template <class F>
__device__ __host__ inline F __m_max(F a, F b) {
    return a > b ? a : b;
}

__device__ __host__ inline AABB merge(const AABB& lhs, const AABB& rhs) noexcept {
    AABB merged;
    merged.upper.x = __m_max(lhs.upper.x, rhs.upper.x);
    merged.upper.y = __m_max(lhs.upper.y, rhs.upper.y);
    merged.upper.z = __m_max(lhs.upper.z, rhs.upper.z);
    merged.lower.x = __m_min(lhs.lower.x, rhs.lower.x);
    merged.lower.y = __m_min(lhs.lower.y, rhs.lower.y);
    merged.lower.z = __m_min(lhs.lower.z, rhs.lower.z);
    return merged;
}

__device__ __host__ inline bool overlap(const AABB& lhs, const AABB& rhs,
                                        const Scalar& gapL) noexcept {
    if ((rhs.lower.x - lhs.upper.x) >= gapL || (lhs.lower.x - rhs.upper.x) >= gapL) return false;
    if ((rhs.lower.y - lhs.upper.y) >= gapL || (lhs.lower.y - rhs.upper.y) >= gapL) return false;
    if ((rhs.lower.z - lhs.upper.z) >= gapL || (lhs.lower.z - rhs.upper.z) >= gapL) return false;
    return true;
}

__device__ __host__ inline Scalar3 centroid(const AABB& box) noexcept {
    Scalar3 c;
    c.x = (box.upper.x + box.lower.x) * 0.5;
    c.y = (box.upper.y + box.lower.y) * 0.5;
    c.z = (box.upper.z + box.lower.z) * 0.5;
    return c;
}

__device__ __host__ inline std::uint32_t expand_bits(std::uint32_t v) noexcept {
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

__device__ __host__ inline std::uint32_t morton_code(Scalar x, Scalar y, Scalar z,
                                                     Scalar resolution = 1024.0) noexcept {
    x = __m_min(__m_max(x * resolution, 0.0), resolution - 1.0);
    y = __m_min(__m_max(y * resolution, 0.0), resolution - 1.0);
    z = __m_min(__m_max(z * resolution, 0.0), resolution - 1.0);

    const std::uint32_t xx = expand_bits(static_cast<std::uint32_t>(x));
    const std::uint32_t yy = expand_bits(static_cast<std::uint32_t>(y));
    const std::uint32_t zz = expand_bits(static_cast<std::uint32_t>(z));

    std::uint32_t mchash = ((xx << 2) + (yy << 1) + zz);

    return mchash;
}

__device__ __host__ void AABB::combines(const Scalar& x, const Scalar& y, const Scalar& z) {
    lower = make_Scalar3(__m_min(lower.x, x), __m_min(lower.y, y), __m_min(lower.z, z));
    upper = make_Scalar3(__m_max(upper.x, x), __m_max(upper.y, y), __m_max(upper.z, z));
}

__device__ __host__ void AABB::combines(const Scalar& x, const Scalar& y, const Scalar& z,
                                        const Scalar& xx, const Scalar& yy, const Scalar& zz) {
    lower = make_Scalar3(__m_min(lower.x, x), __m_min(lower.y, y), __m_min(lower.z, z));
    upper = make_Scalar3(__m_max(upper.x, xx), __m_max(upper.y, yy), __m_max(upper.z, zz));
}

__host__ __device__ void AABB::combines(const AABB& aabb) {
    lower = make_Scalar3(__m_min(lower.x, aabb.lower.x), __m_min(lower.y, aabb.lower.y),
                         __m_min(lower.z, aabb.lower.z));
    upper = make_Scalar3(__m_max(upper.x, aabb.upper.x), __m_max(upper.y, aabb.upper.y),
                         __m_max(upper.z, aabb.upper.z));
}

__host__ __device__ Scalar3 AABB::center() {
    return make_Scalar3((upper.x + lower.x) * 0.5, (upper.y + lower.y) * 0.5,
                        (upper.z + lower.z) * 0.5);
}

__device__ __host__ AABB::AABB() {
    lower = make_Scalar3(1e32, 1e32, 1e32);
    upper = make_Scalar3(-1e32, -1e32, -1e32);
}

__device__ inline int common_upper_bits(const unsigned long long int lhs,
                                        const unsigned long long int rhs) noexcept {
    return ::__clzll(lhs ^ rhs);
}

__device__ inline uint2 determine_range(const uint64_t* node_code, const unsigned int num_leaves,
                                        unsigned int idx) {
    if (idx == 0) {
        return make_uint2(0, num_leaves - 1);
    }

    // determine direction of the range
    const uint64_t self_code = node_code[idx];
    const int L_delta = common_upper_bits(self_code, node_code[idx - 1]);
    const int R_delta = common_upper_bits(self_code, node_code[idx + 1]);
    const int d = (R_delta > L_delta) ? 1 : -1;

    // Compute upper bound for the length of the range

    const int delta_min = __m_min(L_delta, R_delta);
    int l_max = 2;
    int delta = -1;
    int i_tmp = idx + d * l_max;
    if (0 <= i_tmp && i_tmp < num_leaves) {
        delta = common_upper_bits(self_code, node_code[i_tmp]);
    }
    while (delta > delta_min) {
        l_max <<= 1;
        i_tmp = idx + d * l_max;
        delta = -1;
        if (0 <= i_tmp && i_tmp < num_leaves) {
            delta = common_upper_bits(self_code, node_code[i_tmp]);
        }
    }

    // Find the other end by binary search
    int l = 0;
    int t = l_max >> 1;
    while (t > 0) {
        i_tmp = idx + (l + t) * d;
        delta = -1;
        if (0 <= i_tmp && i_tmp < num_leaves) {
            delta = common_upper_bits(self_code, node_code[i_tmp]);
        }
        if (delta > delta_min) {
            l += t;
        }
        t >>= 1;
    }
    unsigned int jdx = idx + l * d;
    if (d < 0) {
        unsigned int temp_jdx = jdx;
        jdx = idx;
        idx = temp_jdx;
    }
    return make_uint2(idx, jdx);
}

__device__ inline unsigned int find_split(const uint64_t* node_code, const unsigned int num_leaves,
                                          const unsigned int first,
                                          const unsigned int last) noexcept {
    const uint64_t first_code = node_code[first];
    const uint64_t last_code = node_code[last];
    if (first_code == last_code) {
        return (first + last) >> 1;
    }
    const int delta_node = common_upper_bits(first_code, last_code);

    // binary search...
    int split = first;
    int stride = last - first;
    do {
        stride = (stride + 1) >> 1;
        const int middle = split + stride;
        if (middle < last) {
            const int delta = common_upper_bits(first_code, node_code[middle]);
            if (delta > delta_node) {
                split = middle;
            }
        }
    } while (stride > 1);

    return split;
}

__device__ int _dType_PT(const Scalar3& v0, const Scalar3& v1, const Scalar3& v2,
                         const Scalar3& v3) {
    Scalar3 basis0 = __MATHUTILS__::__minus(v2, v1);
    Scalar3 basis1 = __MATHUTILS__::__minus(v3, v1);
    Scalar3 basis2 = __MATHUTILS__::__minus(v0, v1);

    const Scalar3 nVec = __MATHUTILS__::__v_vec_cross(basis0, basis1);

    basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
    __MATHUTILS__::Matrix3x3S D, D1, D2;

    __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y, basis1.y, nVec.y,
                                 basis0.z, basis1.z, nVec.z);
    __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y, basis1.y, nVec.y,
                                 basis2.z, basis1.z, nVec.z);
    __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y, basis2.y, nVec.y,
                                 basis0.z, basis2.z, nVec.z);

    Scalar2 param[3];
    param[0].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
    param[0].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

    if (param[0].x > 0 && param[0].x < 1 && param[0].y >= 0) {
        return 3;  // PE v1v2
    } else {
        basis0 = __MATHUTILS__::__minus(v3, v2);
        basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
        basis2 = __MATHUTILS__::__minus(v0, v2);

        __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y, basis1.y, nVec.y,
                                     basis0.z, basis1.z, nVec.z);
        __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y, basis1.y, nVec.y,
                                     basis2.z, basis1.z, nVec.z);
        __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y, basis2.y, nVec.y,
                                     basis0.z, basis2.z, nVec.z);

        param[1].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
        param[1].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

        if (param[1].x > 0.0 && param[1].x < 1.0 && param[1].y >= 0.0) {
            return 4;  // PE v2v3
        } else {
            basis0 = __MATHUTILS__::__minus(v1, v3);
            basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
            basis2 = __MATHUTILS__::__minus(v0, v3);

            __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y, basis1.y, nVec.y,
                                         basis0.z, basis1.z, nVec.z);
            __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y, basis1.y, nVec.y,
                                         basis2.z, basis1.z, nVec.z);
            __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y, basis2.y, nVec.y,
                                         basis0.z, basis2.z, nVec.z);

            param[2].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
            param[2].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

            if (param[2].x > 0.0 && param[2].x < 1.0 && param[2].y >= 0.0) {
                return 5;  // PE v3v1
            } else {
                if (param[0].x <= 0.0 && param[2].x >= 1.0) {
                    return 0;  // PP v1
                } else if (param[1].x <= 0.0 && param[0].x >= 1.0) {
                    return 1;  // PP v2
                } else if (param[2].x <= 0.0 && param[1].x >= 1.0) {
                    return 2;  // PP v3
                } else {
                    return 6;  // PT
                }
            }
        }
    }
}

__device__ int _dType_EE(const Scalar3& v0, const Scalar3& v1, const Scalar3& v2,
                         const Scalar3& v3) {
    Scalar3 u = __MATHUTILS__::__minus(v1, v0);
    Scalar3 v = __MATHUTILS__::__minus(v3, v2);
    Scalar3 w = __MATHUTILS__::__minus(v0, v2);

    Scalar a = __MATHUTILS__::__squaredNorm(u);
    Scalar b = __MATHUTILS__::__v_vec_dot(u, v);
    Scalar c = __MATHUTILS__::__squaredNorm(v);
    Scalar d = __MATHUTILS__::__v_vec_dot(u, w);
    Scalar e = __MATHUTILS__::__v_vec_dot(v, w);

    Scalar D = a * c - b * b;  // always >= 0
    Scalar tD = D;             // tc = tN / tD, default tD = D >= 0
    Scalar sN, tN;
    int defaultCase = 8;
    sN = (b * e - c * d);
    if (sN <= 0.0) {  // sc < 0 => the s=0 edge is visible
        tN = e;
        tD = c;
        defaultCase = 2;
    } else if (sN >= D) {  // sc > 1  => the s=1 edge is visible
        tN = e + b;
        tD = c;
        defaultCase = 5;
    } else {
        tN = (a * e - b * d);
        if (tN > 0.0 && tN < tD &&
            (__MATHUTILS__::__v_vec_dot(w, __MATHUTILS__::__v_vec_cross(u, v)) == 0.0 ||
             __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(u, v)) < 1.0e-20 * a * c)) {
            if (sN < D / 2) {
                tN = e;
                tD = c;
                defaultCase = 2;
            } else {
                tN = e + b;
                tD = c;
                defaultCase = 5;
            }
        }
    }

    if (tN <= 0.0) {
        if (-d <= 0.0) {
            return 0;
        } else if (-d >= a) {
            return 3;
        } else {
            return 6;
        }
    } else if (tN >= tD) {
        if ((-d + b) <= 0.0) {
            return 1;
        } else if ((-d + b) >= a) {
            return 4;
        } else {
            return 7;
        }
    }

    return defaultCase;
}

__device__ inline bool _checkPTintersection(const Scalar3* _vertexes, const uint32_t& id0,
                                            const uint32_t& id1, const uint32_t& id2,
                                            const uint32_t& id3, const Scalar& dHat,
                                            uint32_t* _cpNum, int* _mInx, int4* _collisionPair,
                                            int4* _ccd_collisionPair) noexcept {
    Scalar3 v0 = _vertexes[id0];
    Scalar3 v1 = _vertexes[id1];
    Scalar3 v2 = _vertexes[id2];
    Scalar3 v3 = _vertexes[id3];

    int dtype = _dType_PT(v0, v1, v2, v3);

    Scalar d = 100;
    switch (dtype) {
        case 0: {
            __MATHUTILS__::_d_PP(v0, v1, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, -1, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 2, 1);
            }
            break;
        }

        case 1: {
            __MATHUTILS__::_d_PP(v0, v2, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, -1, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 2, 1);
            }
            break;
        }

        case 2: {
            __MATHUTILS__::_d_PP(v0, v3, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id3, -1, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 2, 1);
            }
            break;
        }

        case 3: {
            __MATHUTILS__::_d_PE(v0, v1, v2, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 3, 1);
            }
            break;
        }

        case 4: {
            __MATHUTILS__::_d_PE(v0, v2, v3, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, id3, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 3, 1);
            }
            break;
        }

        case 5: {
            __MATHUTILS__::_d_PE(v0, v3, v1, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id3, id1, -1);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 3, 1);
            }
            break;
        }

        case 6: {
            __MATHUTILS__::_d_PT(v0, v1, v2, v3, d);
            if (d < dHat) {
                // printf("%d   %d   %d   %d   %d   %f\n", dtype, idx, _faces[obj_idx].x,
                // _faces[obj_idx].y, _faces[obj_idx].z, d);
                int cdp_idx = atomicAdd(_cpNum, 1);
                _ccd_collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                _collisionPair[cdp_idx] = make_int4(-id0 - 1, id1, id2, id3);
                // printf("ccbcbcbcbbcbcbbcbcb  %d  %d  %d  %d\n", -id0 - 1, id1, id2, id3);
                _mInx[cdp_idx] = atomicAdd(_cpNum + 4, 1);
            }
            break;
        }

        default:
            break;
    }
}

__device__ inline bool _checkPTintersection_fullCCD(const Scalar3* _vertexes, const uint32_t& id0,
                                                    const uint32_t& id1, const uint32_t& id2,
                                                    const uint32_t& id3, const Scalar& dHat,
                                                    uint32_t* _cpNum,
                                                    int4* _ccd_collisionPair) noexcept {
    Scalar3 v0 = _vertexes[id0];
    Scalar3 v1 = _vertexes[id1];
    Scalar3 v2 = _vertexes[id2];
    Scalar3 v3 = _vertexes[id3];

    int dtype = _dType_PT(v0, v1, v2, v3);

    Scalar3 basis0 = __MATHUTILS__::__minus(v2, v1);
    Scalar3 basis1 = __MATHUTILS__::__minus(v3, v1);
    Scalar3 basis2 = __MATHUTILS__::__minus(v0, v1);

    const Scalar3 nVec = __MATHUTILS__::__v_vec_cross(basis0, basis1);

    Scalar sign = __MATHUTILS__::__v_vec_dot(nVec, basis2);

    if (dtype == 6 && (sign < 0)) {
        return;
    }

    _ccd_collisionPair[atomicAdd(_cpNum, 1)] = make_int4(-id0 - 1, id1, id2, id3);
}

__device__ inline bool _checkEEintersection(const Scalar3* _vertexes, const Scalar3* _rest_vertexes,
                                            const uint32_t& id0, const uint32_t& id1,
                                            const uint32_t& id2, const uint32_t& id3,
                                            const uint32_t& obj_idx, const Scalar& dHat,
                                            uint32_t* _cpNum, int* MatIndex, int4* _collisionPair,
                                            int4* _ccd_collisionPair, int surfedgeNum) noexcept {
    Scalar3 v0 = _vertexes[id0];
    Scalar3 v1 = _vertexes[id1];
    Scalar3 v2 = _vertexes[id2];
    Scalar3 v3 = _vertexes[id3];

    int dtype = _dType_EE(v0, v1, v2, v3);
    int add_e = -1;
    Scalar d = 100.0;
    bool smooth = true;
    switch (dtype) {
        case 0: {
            __MATHUTILS__::_d_PP(v0, v2, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id0 - 1, -id2 - 1, -id1 - 1, -id3 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);

                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                }
            }
            break;
        }

        case 1: {
            __MATHUTILS__::_d_PP(v0, v3, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id0 - 1, -id3 - 1, -id1 - 1, -id2 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id3, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id3, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                }
            }
            break;
        }

        case 2: {
            __MATHUTILS__::_d_PE(v0, v2, v3, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id0 - 1, -id2 - 1, id3, -id1 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, id3, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id0 - 1, id2, id3, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                }
            }
            break;
        }

        case 3: {
            __MATHUTILS__::_d_PP(v1, v2, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id1 - 1, -id2 - 1, -id0 - 1, -id3 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id2, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id2, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                }
            }
            break;
        }

        case 4: {
            __MATHUTILS__::_d_PP(v1, v3, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id1 - 1, -id3 - 1, -id0 - 1, -id2 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id3, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id3, -1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 2, 1);
                }
            }
            break;
        }

        case 5: {
            __MATHUTILS__::_d_PE(v1, v2, v3, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v0, v1),
                    __MATHUTILS__::__minus(
                        v2,
                        v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                   _rest_vertexes[id2], _rest_vertexes[id3]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id1 - 1, -id2 - 1, id3, -id0 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id2, id3, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id1 - 1, id2, id3, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                }
            }
            break;
        }

        case 6: {
            __MATHUTILS__::_d_PE(v2, v0, v1, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v2, v3),
                    __MATHUTILS__::__minus(
                        v0,
                        v1))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v2, v3))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id2], _rest_vertexes[id3],
                                                   _rest_vertexes[id0], _rest_vertexes[id1]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id2 - 1, -id0 - 1, id1, -id3 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id2 - 1, id0, id1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id2 - 1, id0, id1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                }
            }
            break;
        }

        case 7: {
            __MATHUTILS__::_d_PE(v3, v0, v1, d);
            if (d < dHat) {
                Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                    __MATHUTILS__::__minus(v2, v3),
                    __MATHUTILS__::__minus(
                        v0,
                        v1))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v2, v3))*/;
                Scalar eps_x =
                    __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id2], _rest_vertexes[id3],
                                                   _rest_vertexes[id0], _rest_vertexes[id1]);
                add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

                if (add_e <= -2) {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(-id3 - 1, -id0 - 1, id1, -id2 - 1);
                        MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(-id3 - 1, id0, id1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(-id3 - 1, id0, id1, add_e);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 3, 1);
                }
            }
            break;
        }

        case 8: {
            __MATHUTILS__::_d_EE(v0, v1, v2, v3, d);

            Scalar eeSqureNCross = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
                __MATHUTILS__::__minus(v0, v1),
                __MATHUTILS__::__minus(
                    v2, v3))) /* / __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1))*/;
            Scalar eps_x = __MATHUTILS__::_compute_epx_cp(_rest_vertexes[id0], _rest_vertexes[id1],
                                                          _rest_vertexes[id2], _rest_vertexes[id3]);
            add_e = (eeSqureNCross < eps_x) ? -obj_idx - 2 : -1;

            if (d < dHat) {
                if (add_e <= -2) {
                    // printf("xxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxx\nxxxxxxxxxxx\n");
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    if (smooth) {
                        _collisionPair[cdp_idx] = make_int4(id0, id1, id2, -id3 - 1);
                        break;
                    }
                    _collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                } else {
                    int cdp_idx = atomicAdd(_cpNum, 1);
                    _ccd_collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    _collisionPair[cdp_idx] = make_int4(id0, id1, id2, id3);
                    MatIndex[cdp_idx] = atomicAdd(_cpNum + 4, 1);
                }
            }
            break;
        }

        default:
            break;
    }
}

__global__ void _reduct_max_box(AABB* _leafBoxes, int number) {
    int idof = blockIdx.x * blockDim.x;
    int idx = threadIdx.x + idof;

    extern __shared__ AABB tep[];

    if (idx >= number) return;
    // int cfid = tid + CONFLICT_FREE_OFFSET(tid);
    AABB temp = _leafBoxes[idx];

    __threadfence();

    Scalar xmin = temp.lower.x, ymin = temp.lower.y, zmin = temp.lower.z;
    Scalar xmax = temp.upper.x, ymax = temp.upper.y, zmax = temp.upper.z;
    // printf("%f   %f    %f   %f   %f    %f\n", xmin, ymin, zmin, xmax, ymax, zmax);
    // printf("%f   %f    %f\n", xmax, ymax, zmax);
    int warpTid = threadIdx.x % 32;
    int warpId = (threadIdx.x >> 5);
    int warpNum;
    int tidNum = 32;
    if (blockIdx.x == gridDim.x - 1) {
        warpNum = ((number - idof + 31) >> 5);
        if (warpId == warpNum - 1) {
            tidNum = number - idof - (warpNum - 1) * 32;
        }
    } else {
        warpNum = ((blockDim.x) >> 5);
    }
    for (int i = 1; i < tidNum; i = (i << 1)) {
        temp.combines(__shfl_down_sync(0xFFFFFFFF, xmin, i), __shfl_down_sync(0xFFFFFFFF, ymin, i),
                      __shfl_down_sync(0xFFFFFFFF, zmin, i), __shfl_down_sync(0xFFFFFFFF, xmax, i),
                      __shfl_down_sync(0xFFFFFFFF, ymax, i), __shfl_down_sync(0xFFFFFFFF, zmax, i));
        if (warpTid + i < tidNum) {
            xmin = temp.lower.x, ymin = temp.lower.y, zmin = temp.lower.z;
            xmax = temp.upper.x, ymax = temp.upper.y, zmax = temp.upper.z;
        }
    }
    if (warpTid == 0) {
        tep[warpId] = temp;
    }
    __syncthreads();
    if (threadIdx.x >= warpNum) return;
    if (warpNum > 1) {
        //	tidNum = warpNum;
        temp = tep[threadIdx.x];
        xmin = temp.lower.x, ymin = temp.lower.y, zmin = temp.lower.z;
        xmax = temp.upper.x, ymax = temp.upper.y, zmax = temp.upper.z;
        //	warpNum = ((tidNum + 31) >> 5);
        for (int i = 1; i < warpNum; i = (i << 1)) {
            temp.combines(
                __shfl_down_sync(0xFFFFFFFF, xmin, i), __shfl_down_sync(0xFFFFFFFF, ymin, i),
                __shfl_down_sync(0xFFFFFFFF, zmin, i), __shfl_down_sync(0xFFFFFFFF, xmax, i),
                __shfl_down_sync(0xFFFFFFFF, ymax, i), __shfl_down_sync(0xFFFFFFFF, zmax, i));
            if (threadIdx.x + i < warpNum) {
                xmin = temp.lower.x, ymin = temp.lower.y, zmin = temp.lower.z;
                xmax = temp.upper.x, ymax = temp.upper.y, zmax = temp.upper.z;
            }
        }
    }
    if (threadIdx.x == 0) {
        _leafBoxes[blockIdx.x] = temp;
    }
}

template <class element_type>
__global__ void _calcLeafBvs(const Scalar3* _vertexes, const element_type* _elements, AABB* _bvs,
                             int surffaceNum, int type = 0) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= surffaceNum) return;
    AABB _bv;

    element_type _e = _elements[idx];
    Scalar3 _v = _vertexes[_e.x];
    _bv.combines(_v.x, _v.y, _v.z);
    _v = _vertexes[_e.y];
    _bv.combines(_v.x, _v.y, _v.z);
    if (type == 0) {
        _v = _vertexes[*((uint32_t*)(&_e) + 2)];
        _bv.combines(_v.x, _v.y, _v.z);
    }
    _bvs[idx] = _bv;
}

template <class element_type>
__global__ void _calcLeafBvs_ccd(const Scalar3* _vertexes, const Scalar3* _moveDir, Scalar alpha,
                                 const element_type* _elements, AABB* _bvs, int surffaceNum,
                                 int type = 0) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= surffaceNum) return;
    AABB _bv;

    element_type _e = _elements[idx];
    Scalar3 _v = _vertexes[_e.x];
    Scalar3 _mvD = _moveDir[_e.x];
    _bv.combines(_v.x, _v.y, _v.z);
    _bv.combines(_v.x - _mvD.x * alpha, _v.y - _mvD.y * alpha, _v.z - _mvD.z * alpha);

    _v = _vertexes[_e.y];
    _mvD = _moveDir[_e.y];
    _bv.combines(_v.x, _v.y, _v.z);
    _bv.combines(_v.x - _mvD.x * alpha, _v.y - _mvD.y * alpha, _v.z - _mvD.z * alpha);
    if (type == 0) {
        _v = _vertexes[*((uint32_t*)(&_e) + 2)];
        _mvD = _moveDir[*((uint32_t*)(&_e) + 2)];
        _bv.combines(_v.x, _v.y, _v.z);
        _bv.combines(_v.x - _mvD.x * alpha, _v.y - _mvD.y * alpha, _v.z - _mvD.z * alpha);
    }
    _bvs[idx] = _bv;
}

__global__ void _calcMChash(uint64_t* _MChash, AABB* _bvs, int number) {
    uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= number) return;
    AABB maxBv = _bvs[0];
    Scalar3 SceneSize = make_Scalar3(maxBv.upper.x - maxBv.lower.x, maxBv.upper.y - maxBv.lower.y,
                                     maxBv.upper.z - maxBv.lower.z);
    Scalar3 centerP = _bvs[idx + number - 1].center();
    Scalar3 offset = make_Scalar3(centerP.x - maxBv.lower.x, centerP.y - maxBv.lower.y,
                                  centerP.z - maxBv.lower.z);

    // printf("%d   %f     %f     %f\n", offset.x, offset.y, offset.z);
    uint64_t mc32 =
        morton_code(offset.x / SceneSize.x, offset.y / SceneSize.y, offset.z / SceneSize.z);
    uint64_t mc64 = ((mc32 << 32) | idx);
    _MChash[idx] = mc64;
}

__global__ void _calcLeafNodes(Node* _nodes, const uint32_t* _indices, int number) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= number) return;
    if (idx < number - 1) {
        _nodes[idx].left_idx = 0xFFFFFFFF;
        _nodes[idx].right_idx = 0xFFFFFFFF;
        _nodes[idx].parent_idx = 0xFFFFFFFF;
        _nodes[idx].element_idx = 0xFFFFFFFF;
    }
    int l_idx = idx + number - 1;
    _nodes[l_idx].left_idx = 0xFFFFFFFF;
    _nodes[l_idx].right_idx = 0xFFFFFFFF;
    _nodes[l_idx].parent_idx = 0xFFFFFFFF;
    _nodes[l_idx].element_idx = _indices[idx];
}

__global__ void _calcInternalNodes(Node* _nodes, const uint64_t* _MChash, int number) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= number - 1) return;
    const uint2 ij = determine_range(_MChash, number, idx);
    const unsigned int gamma = find_split(_MChash, number, ij.x, ij.y);

    _nodes[idx].left_idx = gamma;
    _nodes[idx].right_idx = gamma + 1;
    if (__m_min(ij.x, ij.y) == gamma) {
        _nodes[idx].left_idx += number - 1;
    }
    if (__m_max(ij.x, ij.y) == gamma + 1) {
        _nodes[idx].right_idx += number - 1;
    }
    _nodes[_nodes[idx].left_idx].parent_idx = idx;
    _nodes[_nodes[idx].right_idx].parent_idx = idx;
}

__global__ void _calcInternalAABB(const Node* _nodes, AABB* _bvs, uint32_t* flags, int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;
    idx = idx + number - 1;

    uint32_t parent = _nodes[idx].parent_idx;
    while (parent != 0xFFFFFFFF)  // means idx == 0
    {
        const int old = atomicCAS(flags + parent, 0xFFFFFFFF, 0);
        if (old == 0xFFFFFFFF) {
            return;
        }

        const uint32_t lidx = _nodes[parent].left_idx;
        const uint32_t ridx = _nodes[parent].right_idx;

        const AABB lbox = _bvs[lidx];
        const AABB rbox = _bvs[ridx];
        _bvs[parent] = merge(lbox, rbox);

        __threadfence();

        parent = _nodes[parent].parent_idx;
    }
}

__global__ void _sortBvs(const uint32_t* _indices, AABB* _bvs, AABB* _temp_bvs, int number) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= number) return;
    _bvs[idx] = _temp_bvs[_indices[idx]];
}

__global__ void _selfQuery_vf(const int* _btype, const Scalar3* _vertexes, const uint3* _faces,
                              const uint32_t* _surfVerts, const AABB* _bvs, const Node* _nodes,
                              int4* _collisionPair, int4* _ccd_collisionPair, uint32_t* _cpNum,
                              int* MatIndex, Scalar dHat, int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;

    uint32_t stack[64];
    uint32_t* stack_ptr = stack;
    *stack_ptr++ = 0;

    AABB _bv;
    idx = _surfVerts[idx];
    _bv.upper = _vertexes[idx];
    _bv.lower = _vertexes[idx];
    // Scalar bboxDiagSize2 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(_bvs[0].upper,
    // _bvs[0].lower)); printf("%f\n", bboxDiagSize2);
    Scalar gapl = sqrt(dHat);  // 0.001 * sqrt(bboxDiagSize2);
    // Scalar dHat = gapl * gapl;// *bboxDiagSize2;
    unsigned int num_found = 0;
    do {
        const uint32_t node_id = *--stack_ptr;
        const uint32_t L_idx = _nodes[node_id].left_idx;
        const uint32_t R_idx = _nodes[node_id].right_idx;

        if (overlap(_bv, _bvs[L_idx], gapl)) {
            const auto obj_idx = _nodes[L_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (idx != _faces[obj_idx].x && idx != _faces[obj_idx].y &&
                    idx != _faces[obj_idx].z) {
                    if (!(_btype[idx] >= 2 && _btype[_faces[obj_idx].x] >= 2 &&
                          _btype[_faces[obj_idx].y] >= 2 && _btype[_faces[obj_idx].z] >= 2))
                        _checkPTintersection(_vertexes, idx, _faces[obj_idx].x, _faces[obj_idx].y,
                                             _faces[obj_idx].z, dHat, _cpNum, MatIndex,
                                             _collisionPair, _ccd_collisionPair);
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = L_idx;
            }
        }
        if (overlap(_bv, _bvs[R_idx], gapl)) {
            const auto obj_idx = _nodes[R_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (idx != _faces[obj_idx].x && idx != _faces[obj_idx].y &&
                    idx != _faces[obj_idx].z) {
                    if (!(_btype[idx] >= 2 && _btype[_faces[obj_idx].x] >= 2 &&
                          _btype[_faces[obj_idx].y] >= 2 && _btype[_faces[obj_idx].z] >= 2))
                        _checkPTintersection(_vertexes, idx, _faces[obj_idx].x, _faces[obj_idx].y,
                                             _faces[obj_idx].z, dHat, _cpNum, MatIndex,
                                             _collisionPair, _ccd_collisionPair);
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = R_idx;
            }
        }
    } while (stack < stack_ptr);
}

__global__ void _selfQuery_vf_ccd(const int* _btype, const Scalar3* _vertexes,
                                  const Scalar3* moveDir, Scalar alpha, const uint3* _faces,
                                  const uint32_t* _surfVerts, const AABB* _bvs, const Node* _nodes,
                                  int4* _ccd_collisionPair, uint32_t* _cpNum, Scalar dHat,
                                  int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;

    uint32_t stack[64];
    uint32_t* stack_ptr = stack;
    *stack_ptr++ = 0;

    AABB _bv;
    idx = _surfVerts[idx];
    Scalar3 current_vertex = _vertexes[idx];
    Scalar3 mvD = moveDir[idx];
    _bv.upper = current_vertex;
    _bv.lower = current_vertex;
    _bv.combines(current_vertex.x - mvD.x * alpha, current_vertex.y - mvD.y * alpha,
                 current_vertex.z - mvD.z * alpha);
    // Scalar bboxDiagSize2 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(_bvs[0].upper,
    // _bvs[0].lower)); printf("%f\n", bboxDiagSize2);
    Scalar gapl = sqrt(dHat);  // 0.001 * sqrt(bboxDiagSize2);
    // Scalar dHat = gapl * gapl;// *bboxDiagSize2;
    unsigned int num_found = 0;
    do {
        const uint32_t node_id = *--stack_ptr;
        const uint32_t L_idx = _nodes[node_id].left_idx;
        const uint32_t R_idx = _nodes[node_id].right_idx;

        if (overlap(_bv, _bvs[L_idx], gapl)) {
            const auto obj_idx = _nodes[L_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (!(_btype[idx] >= 2 && _btype[_faces[obj_idx].x] >= 2 &&
                      _btype[_faces[obj_idx].y] >= 2 && _btype[_faces[obj_idx].z] >= 2))
                    if (idx != _faces[obj_idx].x && idx != _faces[obj_idx].y &&
                        idx != _faces[obj_idx].z) {
                        _ccd_collisionPair[atomicAdd(_cpNum, 1)] = make_int4(
                            -idx - 1, _faces[obj_idx].x, _faces[obj_idx].y, _faces[obj_idx].z);
                        //_checkPTintersection_fullCCD(_vertexes, idx, _faces[obj_idx].x,
                        //_faces[obj_idx].y, _faces[obj_idx].z, dHat, _cpNum, _ccd_collisionPair);
                    }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = L_idx;
            }
        }
        if (overlap(_bv, _bvs[R_idx], gapl)) {
            const auto obj_idx = _nodes[R_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (!(_btype[idx] >= 2 && _btype[_faces[obj_idx].x] >= 2 &&
                      _btype[_faces[obj_idx].y] >= 2 && _btype[_faces[obj_idx].z] >= 2))
                    if (idx != _faces[obj_idx].x && idx != _faces[obj_idx].y &&
                        idx != _faces[obj_idx].z) {
                        _ccd_collisionPair[atomicAdd(_cpNum, 1)] = make_int4(
                            -idx - 1, _faces[obj_idx].x, _faces[obj_idx].y, _faces[obj_idx].z);
                        //_checkPTintersection_fullCCD(_vertexes, idx, _faces[obj_idx].x,
                        //_faces[obj_idx].y, _faces[obj_idx].z, dHat, _cpNum, _ccd_collisionPair);
                    }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = R_idx;
            }
        }
    } while (stack < stack_ptr);
}

__global__ void _selfQuery_ee(const int* _btype, const Scalar3* _vertexes,
                              const Scalar3* _rest_vertexes, const uint2* _edges, const AABB* _bvs,
                              const Node* _nodes, int4* _collisionPair, int4* _ccd_collisionPair,
                              uint32_t* _cpNum, int* MatIndex, Scalar dHat, int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;

    uint32_t stack[64];
    uint32_t* stack_ptr = stack;
    *stack_ptr++ = 0;

    idx = idx + number - 1;
    AABB _bv = _bvs[idx];
    uint32_t self_eid = _nodes[idx].element_idx;
    // Scalar bboxDiagSize2 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(_bvs[0].upper,
    // _bvs[0].lower)); printf("%f\n", bboxDiagSize2);
    Scalar gapl = sqrt(dHat);  // 0.001 * sqrt(bboxDiagSize2);
    // Scalar dHat = gapl * gapl;// *bboxDiagSize2;
    unsigned int num_found = 0;
    do {
        const uint32_t node_id = *--stack_ptr;
        const uint32_t L_idx = _nodes[node_id].left_idx;
        const uint32_t R_idx = _nodes[node_id].right_idx;

        if (overlap(_bv, _bvs[L_idx], gapl)) {
            const auto obj_idx = _nodes[L_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (self_eid != obj_idx) {
                    if (!(_edges[self_eid].x == _edges[obj_idx].x ||
                          _edges[self_eid].x == _edges[obj_idx].y ||
                          _edges[self_eid].y == _edges[obj_idx].x ||
                          _edges[self_eid].y == _edges[obj_idx].y || obj_idx < self_eid)) {
                        // printf("%d   %d   %d   %d\n", _edges[self_eid].x, _edges[self_eid].y,
                        // _edges[obj_idx].x, _edges[obj_idx].y);
                        if (!(_btype[_edges[self_eid].x] >= 2 && _btype[_edges[self_eid].y] >= 2 &&
                              _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                            _checkEEintersection(_vertexes, _rest_vertexes, _edges[self_eid].x,
                                                 _edges[self_eid].y, _edges[obj_idx].x,
                                                 _edges[obj_idx].y, obj_idx, dHat, _cpNum, MatIndex,
                                                 _collisionPair, _ccd_collisionPair, number);
                    }
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = L_idx;
            }
        }
        if (overlap(_bv, _bvs[R_idx], gapl)) {
            const auto obj_idx = _nodes[R_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (self_eid != obj_idx) {
                    if (!(_edges[self_eid].x == _edges[obj_idx].x ||
                          _edges[self_eid].x == _edges[obj_idx].y ||
                          _edges[self_eid].y == _edges[obj_idx].x ||
                          _edges[self_eid].y == _edges[obj_idx].y || obj_idx < self_eid)) {
                        // printf("%d   %d   %d   %d\n", _edges[self_eid].x, _edges[self_eid].y,
                        // _edges[obj_idx].x, _edges[obj_idx].y);
                        if (!(_btype[_edges[self_eid].x] >= 2 && _btype[_edges[self_eid].y] >= 2 &&
                              _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                            _checkEEintersection(_vertexes, _rest_vertexes, _edges[self_eid].x,
                                                 _edges[self_eid].y, _edges[obj_idx].x,
                                                 _edges[obj_idx].y, obj_idx, dHat, _cpNum, MatIndex,
                                                 _collisionPair, _ccd_collisionPair, number);
                    }
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = R_idx;
            }
        }
    } while (stack < stack_ptr);
}

__global__ void _selfQuery_ee_ccd(const int* _btype, const Scalar3* _vertexes,
                                  const Scalar3* moveDir, Scalar alpha, const uint2* _edges,
                                  const AABB* _bvs, const Node* _nodes, int4* _ccd_collisionPair,
                                  uint32_t* _cpNum, Scalar dHat, int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;

    uint32_t stack[64];
    uint32_t* stack_ptr = stack;
    *stack_ptr++ = 0;
    idx = idx + number - 1;
    AABB _bv = _bvs[idx];
    uint32_t self_eid = _nodes[idx].element_idx;
    uint2 current_edge = _edges[self_eid];
    // Scalar3 edge_tvert0 = __MATHUTILS__::__minus(_vertexes[current_edge.x],
    // __MATHUTILS__::__s_vec_multiply(moveDir[current_edge.x], alpha)); Scalar3 edge_tvert1 =
    // __MATHUTILS__::__minus(_vertexes[current_edge.y],
    // __MATHUTILS__::__s_vec_multiply(moveDir[current_edge.y], alpha)); _bv.combines(edge_tvert0.x,
    //edge_tvert0.y, edge_tvert0.z); _bv.combines(edge_tvert1.x, edge_tvert1.y, edge_tvert1.z);
    Scalar gapl = sqrt(dHat);

    unsigned int num_found = 0;
    do {
        const uint32_t node_id = *--stack_ptr;
        const uint32_t L_idx = _nodes[node_id].left_idx;
        const uint32_t R_idx = _nodes[node_id].right_idx;

        if (overlap(_bv, _bvs[L_idx], gapl)) {
            const auto obj_idx = _nodes[L_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (self_eid != obj_idx) {
                    if (!(_btype[_edges[self_eid].x] >= 2 && _btype[_edges[self_eid].y] >= 2 &&
                          _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                        if (!(current_edge.x == _edges[obj_idx].x ||
                              current_edge.x == _edges[obj_idx].y ||
                              current_edge.y == _edges[obj_idx].x ||
                              current_edge.y == _edges[obj_idx].y || obj_idx < self_eid)) {
                            _ccd_collisionPair[atomicAdd(_cpNum, 1)] =
                                make_int4(current_edge.x, current_edge.y, _edges[obj_idx].x,
                                          _edges[obj_idx].y);
                        }
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = L_idx;
            }
        }
        if (overlap(_bv, _bvs[R_idx], gapl)) {
            const auto obj_idx = _nodes[R_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (self_eid != obj_idx) {
                    if (!(_btype[_edges[self_eid].x] >= 2 && _btype[_edges[self_eid].y] >= 2 &&
                          _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                        if (!(current_edge.x == _edges[obj_idx].x ||
                              current_edge.x == _edges[obj_idx].y ||
                              current_edge.y == _edges[obj_idx].x ||
                              current_edge.y == _edges[obj_idx].y || obj_idx < self_eid)) {
                            _ccd_collisionPair[atomicAdd(_cpNum, 1)] =
                                make_int4(current_edge.x, current_edge.y, _edges[obj_idx].x,
                                          _edges[obj_idx].y);
                        }
                }
            } else  // the node is not a leaf.
            {
                *stack_ptr++ = R_idx;
            }
        }
    } while (stack < stack_ptr);
}

///////////////////////////////////////host//////////////////////////////////////////////

AABB calcMaxBV(AABB* _leafBoxes, AABB* _tempLeafBox, const int& number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;

    unsigned int sharedMsize = sizeof(AABB) * (threadNum >> 5);

    // AABB* _tempLeafBox;
    // CUDA_SAFE_CALL(hipMalloc((void**)&_tempLeafBox, number * sizeof(AABB)));
    CUDA_SAFE_CALL(hipMemcpy(_tempLeafBox, _leafBoxes + number - 1, number * sizeof(AABB),
                              hipMemcpyDeviceToDevice));

    _reduct_max_box<<<blockNum, threadNum, sharedMsize>>>(_tempLeafBox, numbers);

    numbers = blockNum;
    blockNum = (numbers + threadNum - 1) / threadNum;

    while (numbers > 1) {
        _reduct_max_box<<<blockNum, threadNum, sharedMsize>>>(_tempLeafBox, numbers);
        numbers = blockNum;
        blockNum = (numbers + threadNum - 1) / threadNum;
    }
    hipMemcpy(_leafBoxes, _tempLeafBox, sizeof(AABB), hipMemcpyDeviceToDevice);
    AABB h_bv;
    hipMemcpy(&h_bv, _tempLeafBox, sizeof(AABB), hipMemcpyDeviceToHost);
    // CUDA_SAFE_CALL(hipFree(_tempLeafBox));
    return h_bv;
}

template <class element_type>
void calcLeafBvs(const Scalar3* _vertexes, const element_type* _faces, AABB* _bvs,
                 const int& surffaceNum, const int& type) {
    int numbers = surffaceNum;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    _calcLeafBvs<<<blockNum, threadNum>>>(_vertexes, _faces, _bvs + numbers - 1, surffaceNum, type);
}

template <class element_type>
void calcLeafBvs_fullCCD(const Scalar3* _vertexes, const Scalar3* _moveDir, const Scalar& alpha,
                         const element_type* _faces, AABB* _bvs, const int& surffaceNum,
                         const int& type) {
    int numbers = surffaceNum;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    _calcLeafBvs_ccd<<<blockNum, threadNum>>>(_vertexes, _moveDir, alpha, _faces,
                                              _bvs + numbers - 1, surffaceNum, type);
}

void calcMChash(uint64_t* _MChash, AABB* _bvs, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    _calcMChash<<<blockNum, threadNum>>>(_MChash, _bvs, number);
}

void calcLeafNodes(Node* _nodes, const uint32_t* _indices, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    _calcLeafNodes<<<blockNum, threadNum>>>(_nodes, _indices, number);
}

void calcInternalNodes(Node* _nodes, const uint64_t* _MChash, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    _calcInternalNodes<<<blockNum, threadNum>>>(_nodes, _MChash, number);
}

void calcInternalAABB(const Node* _nodes, AABB* _bvs, uint32_t* flags, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    // uint32_t* flags;
    // CUDA_SAFE_CALL(hipMalloc((void**)&flags, (numbers-1) * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemset(flags, 0xFFFFFFFF, sizeof(uint32_t) * (numbers - 1)));
    _calcInternalAABB<<<blockNum, threadNum>>>(_nodes, _bvs, flags, numbers);
    // CUDA_SAFE_CALL(hipFree(flags));
}

void sortBvs(const uint32_t* _indices, AABB* _bvs, AABB* _temp_bvs, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    // AABB* _temp_bvs = _tempLeafBox;
    // CUDA_SAFE_CALL(hipMalloc((void**)&_temp_bvs, (number) * sizeof(AABB)));
    hipMemcpy(_temp_bvs, _bvs + number - 1, sizeof(AABB) * number, hipMemcpyDeviceToDevice);
    _sortBvs<<<blockNum, threadNum>>>(_indices, _bvs + number - 1, _temp_bvs, number);
    // CUDA_SAFE_CALL(hipFree(_temp_bvs));
}

void selfQuery_ee(const int* _btype, const Scalar3* _vertexes, const Scalar3* _rest_vertexes,
                  const uint2* _edges, const AABB* _bvs, const Node* _nodes, int4* _collisionPairs,
                  int4* _ccd_collisionPairs, uint32_t* _cpNum, int* MatIndex, Scalar dHat,
                  int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;

    _selfQuery_ee<<<blockNum, threadNum>>>(_btype, _vertexes, _rest_vertexes, _edges, _bvs, _nodes,
                                           _collisionPairs, _ccd_collisionPairs, _cpNum, MatIndex,
                                           dHat, numbers);
}

void fullCCDselfQuery_ee(const int* _btype, const Scalar3* _vertexes, const Scalar3* moveDir,
                         const Scalar& alpha, const uint2* _edges, const AABB* _bvs,
                         const Node* _nodes, int4* _ccd_collisionPairs, uint32_t* _cpNum,
                         Scalar dHat, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;

    _selfQuery_ee_ccd<<<blockNum, threadNum>>>(_btype, _vertexes, moveDir, alpha, _edges, _bvs,
                                               _nodes, _ccd_collisionPairs, _cpNum, dHat, numbers);
}

void selfQuery_vf(const int* _btype, const Scalar3* _vertexes, const uint3* _faces,
                  const uint32_t* _surfVerts, const AABB* _bvs, const Node* _nodes,
                  int4* _collisionPairs, int4* _ccd_collisionPairs, uint32_t* _cpNum, int* MatIndex,
                  Scalar dHat, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;

    _selfQuery_vf<<<blockNum, threadNum>>>(_btype, _vertexes, _faces, _surfVerts, _bvs, _nodes,
                                           _collisionPairs, _ccd_collisionPairs, _cpNum, MatIndex,
                                           dHat, numbers);
}

void fullCCDselfQuery_vf(const int* _btype, const Scalar3* _vertexes, const Scalar3* moveDir,
                         const Scalar& alpha, const uint3* _faces, const uint32_t* _surfVerts,
                         const AABB* _bvs, const Node* _nodes, int4* _ccd_collisionPairs,
                         uint32_t* _cpNum, Scalar dHat, int number) {
    int numbers = number;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;

    _selfQuery_vf_ccd<<<blockNum, threadNum>>>(_btype, _vertexes, moveDir, alpha, _faces,
                                               _surfVerts, _bvs, _nodes, _ccd_collisionPairs,
                                               _cpNum, dHat, numbers);
}

void LBVH::CUDA_FREE_LBVH() {
    CUDA_SAFE_CALL(hipFree(_indices));
    CUDA_SAFE_CALL(hipFree(_MChash));
    CUDA_SAFE_CALL(hipFree(_nodes));
    CUDA_SAFE_CALL(hipFree(_bvs));
    CUDA_SAFE_CALL(hipFree(_flags));
    CUDA_SAFE_CALL(hipFree(_tempLeafBox));
}

void LBVH::CUDA_MALLOC_LBVH(const int& number) {
    CUDA_SAFE_CALL(hipMalloc((void**)&_indices, (number) * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&_MChash, (number) * sizeof(uint64_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&_nodes, (2 * number - 1) * sizeof(Node)));
    CUDA_SAFE_CALL(hipMalloc((void**)&_bvs, (2 * number - 1) * sizeof(AABB)));
    CUDA_SAFE_CALL(hipMalloc((void**)&_tempLeafBox, number * sizeof(AABB)));
    CUDA_SAFE_CALL(hipMalloc((void**)&_flags, (number - 1) * sizeof(uint32_t)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&_cpNum, sizeof(uint32_t)));ye
    // CUDA_SAFE_CALL(hipMemset(_cpNum, 0, sizeof(uint32_t)));
}

LBVH::~LBVH() {
    // FREE_DEVICE_MEM();
}

void LBVH_E::init(int* _mbtype, Scalar3* _mVerts, Scalar3* _mRestVerts, uint2* _mSurfEdges,
                  int4* _mCollisionPairs, int4* _ccd_mCollisionPairs, uint32_t* _mcpNum,
                  int* _mMatIndex, const int& surfedgeNum, const int& surfvertNum) {
    this->_btype = _mbtype;
    this->_vertexes = _mVerts;
    this->_rest_vertexes = _mRestVerts;
    this->_surfEdges = _mSurfEdges;
    this->_collisionPair = _mCollisionPairs;
    this->_ccd_collisionPair = _ccd_mCollisionPairs;
    this->_cpNum = _mcpNum;
    this->_MatIndex = _mMatIndex;
    this->surfEdgeNum = surfedgeNum;
    this->surfVertNum = surfvertNum;

    CUDA_MALLOC_LBVH(surfEdgeNum);
}

void LBVH_F::init(int* _mbtype, Scalar3* _mVerts, uint3* _mSurfFaces, uint32_t* _mSurfVerts,
                  int4* _mCollisionPairs, int4* _ccd_mCollisionPairs, uint32_t* _mcpNum,
                  int* _mMatIndex, const int& surffaceNum, const int& surfvertNum) {
    this->_btype = _mbtype;
    this->_vertexes = _mVerts;
    this->_surfFaces = _mSurfFaces;
    this->_surfVerts = _mSurfVerts;
    this->_collisionPair = _mCollisionPairs;
    this->_ccd_collisionPair = _ccd_mCollisionPairs;
    this->_cpNum = _mcpNum;
    this->_MatIndex = _mMatIndex;
    this->surfFaceNum = surffaceNum;
    this->surfVertNum = surfvertNum;
    CUDA_MALLOC_LBVH(surfFaceNum);
}

AABB* LBVH_F::getSceneSize() {
    calcLeafBvs(_vertexes, _surfFaces, _bvs, surfFaceNum, 0);

    calcMaxBV(_bvs, _tempLeafBox, surfFaceNum);
    return _bvs;
}

Scalar LBVH_F::Construct() {
    calcLeafBvs(_vertexes, _surfFaces, _bvs, surfFaceNum, 0);
    // CUDA_SAFE_CALL(hipDeviceSynchronize());
    scene = calcMaxBV(_bvs, _tempLeafBox, surfFaceNum);
    calcMChash(_MChash, _bvs, surfFaceNum);
    thrust::sequence(thrust::device_ptr<uint32_t>(_indices),
                     thrust::device_ptr<uint32_t>(_indices) + surfFaceNum);
    thrust::sort_by_key(thrust::device_ptr<uint64_t>(_MChash),
                        thrust::device_ptr<uint64_t>(_MChash) + surfFaceNum,
                        thrust::device_ptr<uint32_t>(_indices));
    sortBvs(_indices, _bvs, _tempLeafBox, surfFaceNum);
    calcLeafNodes(_nodes, _indices, surfFaceNum);
    calcInternalNodes(_nodes, _MChash, surfFaceNum);
    // CUDA_SAFE_CALL(hipDeviceSynchronize());
    calcInternalAABB(_nodes, _bvs, _flags, surfFaceNum);
    return 0;  // time0 + time1 + time2;
}

Scalar LBVH_F::ConstructFullCCD(const Scalar3* moveDir, const Scalar& alpha) {
    calcLeafBvs_fullCCD(_vertexes, moveDir, alpha, _surfFaces, _bvs, surfFaceNum, 0);
    scene = calcMaxBV(_bvs, _tempLeafBox, surfFaceNum);
    calcMChash(_MChash, _bvs, surfFaceNum);
    thrust::sequence(thrust::device_ptr<uint32_t>(_indices),
                     thrust::device_ptr<uint32_t>(_indices) + surfFaceNum);
    thrust::sort_by_key(thrust::device_ptr<uint64_t>(_MChash),
                        thrust::device_ptr<uint64_t>(_MChash) + surfFaceNum,
                        thrust::device_ptr<uint32_t>(_indices));
    sortBvs(_indices, _bvs, _tempLeafBox, surfFaceNum);
    calcLeafNodes(_nodes, _indices, surfFaceNum);
    calcInternalNodes(_nodes, _MChash, surfFaceNum);
    calcInternalAABB(_nodes, _bvs, _flags, surfFaceNum);

    return 0;
}

Scalar LBVH_E::Construct() {
    /*hipEvent_t start, end0, end1, end2;
    hipEventCreate(&start);
    hipEventCreate(&end0);
    hipEventCreate(&end1);
    hipEventCreate(&end2);

    hipEventRecord(start);*/
    calcLeafBvs(_vertexes, _surfEdges, _bvs, surfEdgeNum, 1);
    scene = calcMaxBV(_bvs, _tempLeafBox, surfEdgeNum);
    calcMChash(_MChash, _bvs, surfEdgeNum);
    thrust::sequence(thrust::device_ptr<uint32_t>(_indices),
                     thrust::device_ptr<uint32_t>(_indices) + surfEdgeNum);
    // hipEventRecord(end0);

    thrust::sort_by_key(thrust::device_ptr<uint64_t>(_MChash),
                        thrust::device_ptr<uint64_t>(_MChash) + surfEdgeNum,
                        thrust::device_ptr<uint32_t>(_indices));
    sortBvs(_indices, _bvs, _tempLeafBox, surfEdgeNum);

    // hipEventRecord(end1);

    calcLeafNodes(_nodes, _indices, surfEdgeNum);

    calcInternalNodes(_nodes, _MChash, surfEdgeNum);
    // CUDA_SAFE_CALL(hipDeviceSynchronize());
    calcInternalAABB(_nodes, _bvs, _flags, surfEdgeNum);
    // selfQuery(_vertexes, _edges, _bvs, _nodes, _collisionPair, _cpNum, surfEdgeNum);
    // hipEventRecord(end2);
    // CUDA_SAFE_CALL(hipDeviceSynchronize());
    /*float time0 = 0, time1 = 0, time2 = 0;
    hipEventElapsedTime(&time0, start, end0);
    hipEventElapsedTime(&time1, end0, end1);
    hipEventElapsedTime(&time2, end1, end2);
    (hipEventDestroy(start));
    (hipEventDestroy(end0));
    (hipEventDestroy(end1));
    (hipEventDestroy(end2));*/
    // std::cout << "sort time: " << time1 << std::endl;
    return 0;  // time0 + time1 + time2;
    // std::cout << "generation done: " << time0 + time1 + time2 << std::endl;
}

Scalar LBVH_E::ConstructFullCCD(const Scalar3* moveDir, const Scalar& alpha) {
    calcLeafBvs_fullCCD(_vertexes, moveDir, alpha, _surfEdges, _bvs, surfEdgeNum, 1);
    scene = calcMaxBV(_bvs, _tempLeafBox, surfEdgeNum);
    calcMChash(_MChash, _bvs, surfEdgeNum);
    thrust::sequence(thrust::device_ptr<uint32_t>(_indices),
                     thrust::device_ptr<uint32_t>(_indices) + surfEdgeNum);

    thrust::sort_by_key(thrust::device_ptr<uint64_t>(_MChash),
                        thrust::device_ptr<uint64_t>(_MChash) + surfEdgeNum,
                        thrust::device_ptr<uint32_t>(_indices));
    sortBvs(_indices, _bvs, _tempLeafBox, surfEdgeNum);

    calcLeafNodes(_nodes, _indices, surfEdgeNum);

    calcInternalNodes(_nodes, _MChash, surfEdgeNum);

    calcInternalAABB(_nodes, _bvs, _flags, surfEdgeNum);

    return 0;
}

void LBVH_F::SelfCollisionDetect(Scalar dHat) {
    selfQuery_vf(_btype, _vertexes, _surfFaces, _surfVerts, _bvs, _nodes, _collisionPair,
                 _ccd_collisionPair, _cpNum, _MatIndex, dHat, surfVertNum);
}

void LBVH_E::SelfCollisionDetect(Scalar dHat) {
    selfQuery_ee(_btype, _vertexes, _rest_vertexes, _surfEdges, _bvs, _nodes, _collisionPair,
                 _ccd_collisionPair, _cpNum, _MatIndex, dHat, surfEdgeNum);
}

void LBVH_F::SelfCollisionFullDetect(Scalar dHat, const Scalar3* moveDir, const Scalar& alpha) {
    fullCCDselfQuery_vf(_btype, _vertexes, moveDir, alpha, _surfFaces, _surfVerts, _bvs, _nodes,
                        _ccd_collisionPair, _cpNum, dHat, surfVertNum);
}

void LBVH_E::SelfCollisionFullDetect(Scalar dHat, const Scalar3* moveDir, const Scalar& alpha) {
    fullCCDselfQuery_ee(_btype, _vertexes, moveDir, alpha, _surfEdges, _bvs, _nodes,
                        _ccd_collisionPair, _cpNum, dHat, surfEdgeNum);
}

__device__ bool edgeTriIntersect(const Scalar3& ve0, const Scalar3& ve1, const Scalar3& vt0,
                                 const Scalar3& vt1, const Scalar3& vt2) {
    // printf("check for tri and lines\n");

    __MATHUTILS__::Matrix3x3S coefMtr;
    Scalar3 col0 = __MATHUTILS__::__minus(vt1, vt0);
    Scalar3 col1 = __MATHUTILS__::__minus(vt2, vt0);
    Scalar3 col2 = __MATHUTILS__::__minus(ve0, ve1);

    __MATHUTILS__::__set_Mat_val_column(coefMtr, col0, col1, col2);

    Scalar3 n = __MATHUTILS__::__v_vec_cross(col0, col1);
    if (__MATHUTILS__::__v_vec_dot(n, __MATHUTILS__::__minus(ve0, vt0)) *
            __MATHUTILS__::__v_vec_dot(n, __MATHUTILS__::__minus(ve1, vt0)) >
        0) {
        return false;
    }

    Scalar det = __MATHUTILS__::__Determiant(coefMtr);

    if (abs(det) < 1e-20) {
        return false;
    }

    __MATHUTILS__::Matrix3x3S D1, D2, D3;
    Scalar3 b = __MATHUTILS__::__minus(ve0, vt0);

    __MATHUTILS__::__set_Mat_val_column(D1, b, col1, col2);
    __MATHUTILS__::__set_Mat_val_column(D2, col0, b, col2);
    __MATHUTILS__::__set_Mat_val_column(D3, col0, col1, b);

    Scalar uvt[3];
    uvt[0] = __MATHUTILS__::__Determiant(D1) / det;
    uvt[1] = __MATHUTILS__::__Determiant(D2) / det;
    uvt[2] = __MATHUTILS__::__Determiant(D3) / det;

    if (uvt[0] >= 0.0 && uvt[1] >= 0.0 && uvt[0] + uvt[1] <= 1.0 && uvt[2] >= 0.0 &&
        uvt[2] <= 1.0) {
        return true;
    } else {
        return false;
    }
}

__device__ __host__ inline bool _overlap(const AABB& lhs, const AABB& rhs,
                                         const Scalar& gapL) noexcept {
    if ((rhs.lower.x - lhs.upper.x) >= gapL || (lhs.lower.x - rhs.upper.x) >= gapL) return false;
    if ((rhs.lower.y - lhs.upper.y) >= gapL || (lhs.lower.y - rhs.upper.y) >= gapL) return false;
    if ((rhs.lower.z - lhs.upper.z) >= gapL || (lhs.lower.z - rhs.upper.z) >= gapL) return false;
    return true;
}

__global__ void _CollisionDetectTriEdge(const int* _btype, const Scalar3* _vertexes,
                                        const uint2* _edges, const uint3* _faces,
                                        const AABB* _edge_bvs, const Node* _edge_nodes,
                                        int* _isIntesect, Scalar dHat, int number) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= number) return;

    uint32_t stack[64];
    uint32_t* stack_ptr = stack;
    *stack_ptr++ = 0;

    uint3 face = _faces[idx];
    // idx = idx + number - 1;

    AABB _bv;

    Scalar3 _v = _vertexes[face.x];
    _bv.combines(_v.x, _v.y, _v.z);
    _v = _vertexes[face.y];
    _bv.combines(_v.x, _v.y, _v.z);
    _v = _vertexes[face.z];
    _bv.combines(_v.x, _v.y, _v.z);

    // uint32_t self_eid = _edge_nodes[idx].element_idx;
    // Scalar instance->bboxDiagSize2 =
    // __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(_edge_bvs[0].upper, _edge_bvs[0].lower));
    // printf("%f\n", instance->bboxDiagSize2);
    Scalar gapl = 0;  // sqrt(dHat);
    // Scalar dHat = gapl * gapl;// *instance->bboxDiagSize2;

    do {
        const uint32_t node_id = *--stack_ptr;
        const uint32_t L_idx = _edge_nodes[node_id].left_idx;
        const uint32_t R_idx = _edge_nodes[node_id].right_idx;

        if (_overlap(_bv, _edge_bvs[L_idx], gapl)) {
            const auto obj_idx = _edge_nodes[L_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (!(face.x == _edges[obj_idx].x || face.x == _edges[obj_idx].y ||
                      face.y == _edges[obj_idx].x || face.y == _edges[obj_idx].y ||
                      face.z == _edges[obj_idx].x || face.z == _edges[obj_idx].y)) {
                    if (!(_btype[face.x] >= 2 && _btype[face.y] >= 2 && _btype[face.z] >= 2 &&
                          _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                        if (edgeTriIntersect(_vertexes[_edges[obj_idx].x],
                                             _vertexes[_edges[obj_idx].y], _vertexes[face.x],
                                             _vertexes[face.y], _vertexes[face.z])) {
                            // atomicAdd(_isIntesect, -1);
                            *_isIntesect = -1;
                            printf("tri-edge intersection error\n tri: %d %d %d,  edge: %d  %d\n",
                                   face.x, face.y, face.z, _edges[obj_idx].x, _edges[obj_idx].y);
                            return;
                        }
                }

            } else  // the node is not a leaf.
            {
                *stack_ptr++ = L_idx;
            }
        }
        if (_overlap(_bv, _edge_bvs[R_idx], gapl)) {
            const auto obj_idx = _edge_nodes[R_idx].element_idx;
            if (obj_idx != 0xFFFFFFFF) {
                if (!(face.x == _edges[obj_idx].x || face.x == _edges[obj_idx].y ||
                      face.y == _edges[obj_idx].x || face.y == _edges[obj_idx].y ||
                      face.z == _edges[obj_idx].x || face.z == _edges[obj_idx].y)) {
                    if (!(_btype[face.x] >= 2 && _btype[face.y] >= 2 && _btype[face.z] >= 2 &&
                          _btype[_edges[obj_idx].x] >= 2 && _btype[_edges[obj_idx].y] >= 2))
                        if (edgeTriIntersect(_vertexes[_edges[obj_idx].x],
                                             _vertexes[_edges[obj_idx].y], _vertexes[face.x],
                                             _vertexes[face.y], _vertexes[face.z])) {
                            // atomicAdd(_isIntesect, -1);
                            *_isIntesect = -1;
                            printf("tri-edge intersection error\n tri: %d %d %d,  edge: %d  %d\n",
                                   face.x, face.y, face.z, _edges[obj_idx].x, _edges[obj_idx].y);
                            return;
                        }
                }

            } else  // the node is not a leaf.
            {
                *stack_ptr++ = R_idx;
            }
        }
    } while (stack < stack_ptr);
}

void LBVH_EF::init(int* _btype, Scalar3* _mVerts, Scalar3* _rest_vertexes, uint2* _mSurfEdges,
                   uint3* _mSurfFaces, uint32_t* _mSurfVerts, int4* _mCollisionPairs,
                   int4* _ccd_mCollisionPairs, uint32_t* _mcpNum, int* _mMatIndex,
                   AABB* _surfEdge_bvs, Node* _surfEdge_nodes, const int& surfEdgeNum,
                   const int& surfFaceNum, const int& surfVertNum) {
    // 初始化基类 LBVH
    this->_btype = _btype;
    this->_vertexes = _mVerts;
    this->_rest_vertexes = _rest_vertexes;
    this->_surfEdges = _mSurfEdges;
    this->_surfFaces = _mSurfFaces;
    this->_surfVerts = _mSurfVerts;
    this->_collisionPair = _mCollisionPairs;
    this->_ccd_collisionPair = _ccd_mCollisionPairs;
    this->_cpNum = _mcpNum;
    this->_MatIndex = _mMatIndex;
    this->surfEdgeNum = surfEdgeNum;
    this->surfFaceNum = surfFaceNum;
    this->surfVertNum = surfVertNum;
    this->_surfEdge_bvs = _surfEdge_bvs;
    this->_surfEdge_nodes = _surfEdge_nodes;
}

bool LBVH_EF::CollisionDetectTriEdge(Scalar dHat) {
    int numbers = surfFaceNum;
    const unsigned int threadNum = DEFAULT_THREADS;
    int blockNum = (numbers + threadNum - 1) / threadNum;
    int* _isIntersect;
    CUDA_SAFE_CALL(hipMalloc((void**)&_isIntersect, sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(_isIntersect, 0, sizeof(int)));

    _CollisionDetectTriEdge<<<blockNum, threadNum>>>(_btype, _vertexes, _surfEdges, _surfFaces,
                                                     _surfEdge_bvs, _surfEdge_nodes, _isIntersect,
                                                     dHat, numbers);

    int h_isITST;
    hipMemcpy(&h_isITST, _isIntersect, sizeof(int), hipMemcpyDeviceToHost);
    CUDA_SAFE_CALL(hipFree(_isIntersect));
    if (h_isITST < 0) {
        return true;
    }
    return false;
}

bool LBVH_EF::checkCollisionDetectTriEdge(Scalar dHat) { return CollisionDetectTriEdge(dHat); }

void LBVHCollisionDetector::initBVH(std::unique_ptr<GeometryManager>& instance, int* _btype) {
    // init lbvh e
    lbvh_e.init(_btype, instance->getCudaVertPos(), instance->getCudaRestVertPos(),
                instance->getCudaSurfEdge(), instance->getCudaCollisionPairs(),
                instance->getCudaCCDCollisionPairs(), instance->getCudaCPNum(),
                instance->getCudaMatIndex(), instance->getHostNumSurfEdges(),
                instance->getHostNumSurfVerts());

    // init lbvh f
    lbvh_f.init(_btype, instance->getCudaVertPos(), instance->getCudaSurfFace(),
                instance->getCudaSurfVert(), instance->getCudaCollisionPairs(),
                instance->getCudaCCDCollisionPairs(), instance->getCudaCPNum(),
                instance->getCudaMatIndex(), instance->getHostNumSurfFaces(),
                instance->getHostNumSurfVerts());

    // init lbvh_ef
    lbvh_ef.init(_btype, instance->getCudaVertPos(), instance->getCudaRestVertPos(),
                 instance->getCudaSurfEdge(), instance->getCudaSurfFace(),
                 instance->getCudaSurfVert(), instance->getCudaCollisionPairs(),
                 instance->getCudaCCDCollisionPairs(), instance->getCudaCPNum(),
                 instance->getCudaMatIndex(), lbvh_e._bvs, lbvh_e._nodes,
                 instance->getHostNumSurfEdges(), instance->getHostNumSurfFaces(),
                 instance->getHostNumSurfVerts());
}

void LBVHCollisionDetector::buildBVH(std::unique_ptr<GeometryManager>& instance) {
    lbvh_f.Construct();
    lbvh_e.Construct();
}

void LBVHCollisionDetector::buildCP(std::unique_ptr<GeometryManager>& instance) {
    CUDA_SAFE_CALL(hipMemset(instance->getCudaCPNum(), 0, 5 * sizeof(uint32_t)));
    lbvh_f.SelfCollisionDetect(instance->getHostDHat());
    lbvh_e.SelfCollisionDetect(instance->getHostDHat());
    CUDA_SAFE_CALL(hipMemcpy(&instance->getHostCpNum(0), instance->getCudaCPNum(),
                              5 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

void LBVHCollisionDetector::buildBVH_FULLCCD(std::unique_ptr<GeometryManager>& instance,
                                             const Scalar& alpha) {
    lbvh_f.ConstructFullCCD(instance->getCudaMoveDir(), alpha);
    lbvh_e.ConstructFullCCD(instance->getCudaMoveDir(), alpha);
}

void LBVHCollisionDetector::buildFullCP(std::unique_ptr<GeometryManager>& instance,
                                        const Scalar& alpha) {
    CUDA_SAFE_CALL(hipMemset(instance->getCudaCPNum(), 0, sizeof(uint32_t)));

    lbvh_f.SelfCollisionFullDetect(instance->getHostDHat(), instance->getCudaMoveDir(), alpha);
    lbvh_e.SelfCollisionFullDetect(instance->getHostDHat(), instance->getCudaMoveDir(), alpha);

    CUDA_SAFE_CALL(hipMemcpy(&instance->getHostCcdCpNum(), instance->getCudaCPNum(),
                              sizeof(uint32_t), hipMemcpyDeviceToHost));
}

// #include <cstdio>
// #include <cstdlib>
// #include <vector>
//
// #include <hip/hip_runtime.h>
// #include <hipsolver.h>
// #include <random>
//
// #include <cstdlib>
//
// int main2() {
//     hipsolverHandle_t cusolverH = NULL;
//     hipStream_t stream = NULL;
//
//     const int m = 12;
//     const int lda = m;
//     /*
//      *       | 3.5 0.5 0.0 |
//      *   A = | 0.5 3.5 0.0 |
//      *       | 0.0 0.0 2.0 |
//      *
//      */
//     std::vector<Scalar> A;// = { 3.5, 0.5, 0.0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0 };
//     //const std::vector<Scalar> lambda = { 2.0, 3.0, 4.0 };
//     for (int i = 0;i < m;i++) {
//         for (int j = 0;j < m;j++) {
//             A.push_back((Scalar)rand() / RAND_MAX);
//         }
//     }
//
//     std::vector<Scalar> V(lda * m, 0); // eigenvectors
//     std::vector<Scalar> W(m, 0);       // eigenvalues
//
//     Scalar* d_A = nullptr;
//     Scalar* d_W = nullptr;
//     int* d_info = nullptr;
//
//     int info = 0;
//
//     int lwork = 0;            /* size of workspace */
//     Scalar* d_work = nullptr; /* device workspace*/
//
//     std::printf("A = (matlab base-1)\n");
//     //print_matrix(m, m, A.data(), lda);
//     std::printf("=====\n");
//
//     hipEvent_t start, end0;
//     hipEventCreate(&start);
//     hipEventCreate(&end0);
//
//
//     /* step 1: create cusolver handle, bind a stream */
//     (hipsolverDnCreate(&cusolverH));
//
//     (hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
//     (hipsolverSetStream(cusolverH, stream));
//
//     (hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(Scalar) * A.size()));
//     (hipMalloc(reinterpret_cast<void**>(&d_W), sizeof(Scalar) * W.size()));
//     (hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int)));
//
//     (
//         hipMemcpyAsync(d_A, A.data(), sizeof(Scalar) * A.size(), hipMemcpyHostToDevice,
//         stream));
//
//     // step 3: query working space of syevd
//     hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
//     hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
//     hipEventRecord(start);
//     (hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork));
//
//     (hipMalloc(reinterpret_cast<void**>(&d_work), sizeof(Scalar) * lwork));
//
//     // step 4: compute spectrum
//     (
//         hipsolverDnDsyevd(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, d_info));
//     hipEventRecord(end0);
//     (
//         hipMemcpyAsync(V.data(), d_A, sizeof(Scalar) * V.size(), hipMemcpyDeviceToHost,
//         stream));
//     (
//         hipMemcpyAsync(W.data(), d_W, sizeof(Scalar) * W.size(), hipMemcpyDeviceToHost,
//         stream));
//     (hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
//
//     (hipStreamSynchronize(stream));
//
//
//
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//
//     float time0 = 0, time1 = 0, time2 = 0;
//     hipEventElapsedTime(&time0, start, end0);
//
//     (hipEventDestroy(start));
//     (hipEventDestroy(end0));
//
//     std::printf("after syevd: info = %d  %f\n", info, time0);
//     if (0 > info) {
//         std::printf("%d-th parameter is wrong \n", -info);
//         exit(1);
//     }
//
//     std::printf("eigenvalue = (matlab base-1), ascending order\n");
//     int idx = 1;
//     for (auto const& i : W) {
//         std::printf("W[%i] = %E\n", idx, i);
//         idx++;
//     }
//
//
//     (hipFree(d_A));
//     (hipFree(d_W));
//     (hipFree(d_info));
//     (hipFree(d_work));
//
//     (hipsolverDnDestroy(cusolverH));
//
//     (hipStreamDestroy(stream));
//
//     (hipDeviceReset());
//
//     return EXIT_SUCCESS;
// }
