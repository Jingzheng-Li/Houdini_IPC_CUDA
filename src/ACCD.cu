#include "hip/hip_runtime.h"


#include "ACCD.cuh"

namespace __ACCD__ {

__device__ int _dType_point_triangle(const Scalar3& v0, const Scalar3& v1,
                                     const Scalar3& v2, const Scalar3& v3) {
    Scalar3 basis0 = __MATHUTILS__::__minus(v2, v1);
    Scalar3 basis1 = __MATHUTILS__::__minus(v3, v1);
    Scalar3 basis2 = __MATHUTILS__::__minus(v0, v1);

    const Scalar3 nVec = __MATHUTILS__::__v_vec_cross(basis0, basis1);

    basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
    __MATHUTILS__::Matrix3x3S D, D1, D2;

    __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y, basis1.y,
                                 nVec.y, basis0.z, basis1.z, nVec.z);
    __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y, basis1.y,
                                 nVec.y, basis2.z, basis1.z, nVec.z);
    __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y, basis2.y,
                                 nVec.y, basis0.z, basis2.z, nVec.z);

    Scalar2 param[3];
    param[0].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
    param[0].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

    if (param[0].x > 0 && param[0].x < 1 && param[0].y >= 0) {
        return 3;  // PE v1v2
    } else {
        basis0 = __MATHUTILS__::__minus(v3, v2);
        basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
        basis2 = __MATHUTILS__::__minus(v0, v2);

        __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y, basis1.y,
                                     nVec.y, basis0.z, basis1.z, nVec.z);
        __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y, basis1.y,
                                     nVec.y, basis2.z, basis1.z, nVec.z);
        __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y, basis2.y,
                                     nVec.y, basis0.z, basis2.z, nVec.z);

        param[1].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
        param[1].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

        if (param[1].x > 0.0 && param[1].x < 1.0 && param[1].y >= 0.0) {
            return 4;  // PE v2v3
        } else {
            basis0 = __MATHUTILS__::__minus(v1, v3);
            basis1 = __MATHUTILS__::__v_vec_cross(basis0, nVec);
            basis2 = __MATHUTILS__::__minus(v0, v3);

            __MATHUTILS__::__set_Mat_val(D, basis0.x, basis1.x, nVec.x, basis0.y,
                                         basis1.y, nVec.y, basis0.z, basis1.z, nVec.z);
            __MATHUTILS__::__set_Mat_val(D1, basis2.x, basis1.x, nVec.x, basis2.y,
                                         basis1.y, nVec.y, basis2.z, basis1.z, nVec.z);
            __MATHUTILS__::__set_Mat_val(D2, basis0.x, basis2.x, nVec.x, basis0.y,
                                         basis2.y, nVec.y, basis0.z, basis2.z, nVec.z);

            param[2].x = __MATHUTILS__::__Determiant(D1) / __MATHUTILS__::__Determiant(D);
            param[2].y = __MATHUTILS__::__Determiant(D2) / __MATHUTILS__::__Determiant(D);

            if (param[2].x > 0.0 && param[2].x < 1.0 && param[2].y >= 0.0) {
                return 5;  // PE v3v1
            } else {
                if (param[0].x <= 0.0 && param[2].x >= 1.0) {
                    return 0;  // PP v1
                } else if (param[1].x <= 0.0 && param[0].x >= 1.0) {
                    return 1;  // PP v2
                } else if (param[2].x <= 0.0 && param[1].x >= 1.0) {
                    return 2;  // PP v3
                } else {
                    return 6;  // PT
                }
            }
        }
    }
}

__device__ int _dType_edge_edge(const Scalar3& v0, const Scalar3& v1, const Scalar3& v2,
                                const Scalar3& v3) {
    Scalar3 u = __MATHUTILS__::__minus(v1, v0);
    Scalar3 v = __MATHUTILS__::__minus(v3, v2);
    Scalar3 w = __MATHUTILS__::__minus(v0, v2);

    Scalar a = __MATHUTILS__::__squaredNorm(u);
    Scalar b = __MATHUTILS__::__v_vec_dot(u, v);
    Scalar c = __MATHUTILS__::__squaredNorm(v);
    Scalar d = __MATHUTILS__::__v_vec_dot(u, w);
    Scalar e = __MATHUTILS__::__v_vec_dot(v, w);

    Scalar D = a * c - b * b;  // always >= 0
    Scalar tD = D;             // tc = tN / tD, default tD = D >= 0
    Scalar sN, tN;
    int defaultCase = 8;
    sN = (b * e - c * d);
    if (sN <= 0.0) {  // sc < 0 => the s=0 edge is visible
        tN = e;
        tD = c;
        defaultCase = 2;
    } else if (sN >= D) {  // sc > 1  => the s=1 edge is visible
        tN = e + b;
        tD = c;
        defaultCase = 5;
    } else {
        tN = (a * e - b * d);
        if (tN > 0.0 && tN < tD &&
            (__MATHUTILS__::__v_vec_dot(w, __MATHUTILS__::__v_vec_cross(u, v)) == 0.0 ||
             __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(u, v)) <
                 1.0e-20 * a * c)) {
            if (sN < D / 2) {
                tN = e;
                tD = c;
                defaultCase = 2;
            } else {
                tN = e + b;
                tD = c;
                defaultCase = 5;
            }
        }
    }

    if (tN <= 0.0) {
        if (-d <= 0.0) {
            return 0;
        } else if (-d >= a) {
            return 3;
        } else {
            return 6;
        }
    } else if (tN >= tD) {
        if ((-d + b) <= 0.0) {
            return 1;
        } else if ((-d + b) >= a) {
            return 4;
        } else {
            return 7;
        }
    }

    return defaultCase;
}

__device__ __forceinline__ Scalar point_point_distance(const Scalar3& v0,
                                                       const Scalar3& v1) {
    return __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v0, v1));
}

__device__ __forceinline__ Scalar point_triangle_distance(const Scalar3& v0,
                                                          const Scalar3& v1,
                                                          const Scalar3& v2,
                                                          const Scalar3& v3) {
    Scalar3 b = __MATHUTILS__::__v_vec_cross(__MATHUTILS__::__minus(v2, v1),
                                             __MATHUTILS__::__minus(v3, v1));
    // Scalar3 test = __MATHUTILS__::__minus(v0, v1);
    Scalar aTb = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__minus(v0, v1),
                                            b);  //(v0 - v1).dot(b);
    // printf("%f   %f   %f          %f   %f   %f   %f\n", b.x, b.y, b.z, test.x, test.y,
    // test.z, aTb);
    return aTb * aTb / __MATHUTILS__::__squaredNorm(b);
}

__device__ __forceinline__ Scalar point_edge_distance(const Scalar3& v0,
                                                      const Scalar3& v1,
                                                      const Scalar3& v2) {
    return __MATHUTILS__::__squaredNorm(__MATHUTILS__::__v_vec_cross(
               __MATHUTILS__::__minus(v1, v0), __MATHUTILS__::__minus(v2, v0))) /
           __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(v2, v1));
}

__device__ __forceinline__ Scalar edge_edge_distance(const Scalar3& v0, const Scalar3& v1,
                                                     const Scalar3& v2,
                                                     const Scalar3& v3) {
    Scalar3 b = __MATHUTILS__::__v_vec_cross(
        __MATHUTILS__::__minus(v1, v0),
        __MATHUTILS__::__minus(v3, v2));  //(v1 - v0).cross(v3 - v2);
    // if(__MATHUTILS__::__norm(b) <1e-6)
    //     b =
    //     __MATHUTILS__::__v_vec_cross(__MATHUTILS__::__v_vec_cross(__MATHUTILS__::__minus(v1,
    //     v0), __MATHUTILS__::__minus(v2, v0)), __MATHUTILS__::__minus(v1, v0));
    Scalar aTb = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__minus(v2, v0),
                                            b);  //(v2 - v0).dot(b);
    return aTb * aTb / __MATHUTILS__::__squaredNorm(b);
}

__device__ __forceinline__ Scalar _d_EEParallel(const Scalar3& v0, const Scalar3& v1,
                                                const Scalar3& v2, const Scalar3& v3) {
    Scalar3 b = __MATHUTILS__::__v_vec_cross(
        __MATHUTILS__::__v_vec_cross(__MATHUTILS__::__minus(v1, v0),
                                     __MATHUTILS__::__minus(v2, v0)),
        __MATHUTILS__::__minus(v1, v0));
    Scalar aTb = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__minus(v2, v0),
                                            b);  //(v2 - v0).dot(b);
    return aTb * aTb / __MATHUTILS__::__squaredNorm(b);
}

__device__ Scalar edge_edge_distance_unclassified(const Scalar3& ea0, const Scalar3& ea1,
                                                  const Scalar3& eb0,
                                                  const Scalar3& eb1) {
    switch (_dType_edge_edge(ea0, ea1, eb0, eb1)) {
        case 0:
            return point_point_distance(ea0, eb0);
        case 1:
            return point_point_distance(ea0, eb1);
        case 2:
            return point_edge_distance(ea0, eb0, eb1);
        case 3:
            return point_point_distance(ea1, eb0);
        case 4:
            return point_point_distance(ea1, eb1);
        case 5:
            return point_edge_distance(ea1, eb0, eb1);
        case 6:
            return point_edge_distance(eb0, ea0, ea1);
        case 7:
            return point_edge_distance(eb1, ea0, ea1);
        case 8:
            return edge_edge_distance(ea0, ea1, eb0, eb1);
        default:
            return 1e32;
    }
}

__device__ Scalar point_triangle_distance_unclassified(const Scalar3& p,
                                                       const Scalar3& t0,
                                                       const Scalar3& t1,
                                                       const Scalar3& t2) {
    switch (_dType_point_triangle(p, t0, t1, t2)) {
        case 0:
            return point_point_distance(p, t0);
        case 1:
            return point_point_distance(p, t1);
        case 2:
            return point_point_distance(p, t2);
        case 3:
            return point_edge_distance(p, t0, t1);
        case 4:
            return point_edge_distance(p, t1, t2);
        case 5:
            return point_edge_distance(p, t2, t0);
        case 6:
            return point_triangle_distance(p, t0, t1, t2);
        default:
            return 1e32;
    }
}

__device__ Scalar edge_edge_ccd(const Scalar3& _ea0, const Scalar3& _ea1,
                                const Scalar3& _eb0, const Scalar3& _eb1,
                                const Scalar3& _dea0, const Scalar3& _dea1,
                                const Scalar3& _deb0, const Scalar3& _deb1, Scalar eta,
                                Scalar thickness) {
    Scalar3 ea0 = _ea0, ea1 = _ea1, eb0 = _eb0, eb1 = _eb1, dea0 = _dea0, dea1 = _dea1,
            deb0 = _deb0, deb1 = _deb1;
    Scalar3 temp0 = __MATHUTILS__::__add(dea0, dea1);
    Scalar3 temp1 = __MATHUTILS__::__add(deb0, deb1);
    Scalar3 mov =
        __MATHUTILS__::__s_vec_multiply(__MATHUTILS__::__add(temp0, temp1), -0.25);

    dea0 = __MATHUTILS__::__add(dea0, mov);
    dea1 = __MATHUTILS__::__add(dea1, mov);
    deb0 = __MATHUTILS__::__add(deb0, mov);
    deb1 = __MATHUTILS__::__add(deb1, mov);

    Scalar max_disp_mag =
        sqrt(__MATHUTILS__::__m_max(__MATHUTILS__::__squaredNorm(dea0),
                                    __MATHUTILS__::__squaredNorm(dea1))) +
        sqrt(__MATHUTILS__::__m_max(__MATHUTILS__::__squaredNorm(deb0),
                                    __MATHUTILS__::__squaredNorm(deb1)));
    if (max_disp_mag == 0) return 1.0;

    Scalar dist2_cur = edge_edge_distance_unclassified(ea0, ea1, eb0, eb1);

    Scalar dFunc = dist2_cur - thickness * thickness;
    if (dFunc <= 0) {
        Scalar dists0 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea0, eb0));
        Scalar dists1 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea0, eb1));
        Scalar dists2 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea1, eb0));
        Scalar dists3 = __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea1, eb1));

        dist2_cur = __MATHUTILS__::__m_min(__MATHUTILS__::__m_min(dists0, dists1),
                                           __MATHUTILS__::__m_min(dists2, dists3));
        dFunc = dist2_cur - thickness * thickness;
    }
    Scalar dist_cur = sqrt(dist2_cur);
    Scalar gap = eta * dFunc / (dist_cur + thickness);
    Scalar toc = 0.0;
    int count = 0;
    while (true) {
        count++;
        if (count > 50000) return toc;
        // if (count > 5000)
        //     printf("ee  %f  %f  %f\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n\n %f  %f
        //     %f\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n\n\n", _dea0.x, _dea0.y, _dea0.z,
        //     _dea1.x, _dea1.y, _dea1.z,
        //         _deb0.x, _deb0.y, _deb0.z, _deb1.x, _deb1.y, _deb1.z, _ea0.x, _ea0.y,
        //         _ea0.z, _ea1.x, _ea1.y, _ea1.z, _eb0.x, _eb0.y, _eb0.z, _eb1.x, _eb1.y,
        //         _eb1.z);
        Scalar toc_lower_bound =
            (1 - eta) * dFunc / ((dist_cur + thickness) * max_disp_mag);
        ea0 = __MATHUTILS__::__add(
            ea0, __MATHUTILS__::__s_vec_multiply(dea0, toc_lower_bound));
        ea1 = __MATHUTILS__::__add(
            ea1, __MATHUTILS__::__s_vec_multiply(dea1, toc_lower_bound));
        eb0 = __MATHUTILS__::__add(
            eb0, __MATHUTILS__::__s_vec_multiply(deb0, toc_lower_bound));
        eb1 = __MATHUTILS__::__add(
            eb1, __MATHUTILS__::__s_vec_multiply(deb1, toc_lower_bound));

        dist2_cur = edge_edge_distance_unclassified(ea0, ea1, eb0, eb1);
        dFunc = dist2_cur - thickness * thickness;
        if (dFunc <= 0) {
            Scalar dists0 =
                __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea0, eb0));
            Scalar dists1 =
                __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea0, eb1));
            Scalar dists2 =
                __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea1, eb0));
            Scalar dists3 =
                __MATHUTILS__::__squaredNorm(__MATHUTILS__::__minus(ea1, eb1));

            dist2_cur = __MATHUTILS__::__m_min(__MATHUTILS__::__m_min(dists0, dists1),
                                               __MATHUTILS__::__m_min(dists2, dists3));
            dFunc = dist2_cur - thickness * thickness;
        }
        dist_cur = sqrt(dist2_cur);
        if (toc && (dFunc / (dist_cur + thickness) < gap)) {
            break;
        }
        toc += toc_lower_bound;
        if (toc > 1.0) return 1.0;
    }
    return toc;
}

__device__ Scalar point_triangle_ccd(const Scalar3& _p, const Scalar3& _t0,
                                     const Scalar3& _t1, const Scalar3& _t2,
                                     const Scalar3& _dp, const Scalar3& _dt0,
                                     const Scalar3& _dt1, const Scalar3& _dt2, Scalar eta,
                                     Scalar thickness) {
    Scalar3 p = _p, t0 = _t0, t1 = _t1, t2 = _t2, dp = _dp, dt0 = _dt0, dt1 = _dt1,
            dt2 = _dt2;

    Scalar3 temp0 = __MATHUTILS__::__add(dt0, dt1);
    Scalar3 temp1 = __MATHUTILS__::__add(dt2, dp);
    Scalar3 mov =
        __MATHUTILS__::__s_vec_multiply(__MATHUTILS__::__add(temp0, temp1), -0.25);

    dt0 = __MATHUTILS__::__add(dt0, mov);
    dt1 = __MATHUTILS__::__add(dt1, mov);
    dt2 = __MATHUTILS__::__add(dt2, mov);
    dp = __MATHUTILS__::__add(dp, mov);

    Scalar disp_mag2_vec0 = __MATHUTILS__::__squaredNorm(dt0);
    Scalar disp_mag2_vec1 = __MATHUTILS__::__squaredNorm(dt1);
    Scalar disp_mag2_vec2 = __MATHUTILS__::__squaredNorm(dt2);

    Scalar max_disp_mag =
        __MATHUTILS__::__norm(dp) +
        sqrt(__MATHUTILS__::__m_max(
            disp_mag2_vec0, __MATHUTILS__::__m_max(disp_mag2_vec1, disp_mag2_vec2)));
    if (max_disp_mag == 0) return 1.0;

    Scalar dist2_cur = point_triangle_distance_unclassified(p, t0, t1, t2);
    Scalar dist_cur = sqrt(dist2_cur);
    Scalar gap = eta * (dist2_cur - thickness * thickness) / (dist_cur + thickness);
    Scalar toc = 0.0;
    int count = 0;
    while (true) {
        count++;
        if (count > 50000) return toc;
        // if (count > 5000)
        //     printf("pt  %f  %f  %f\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n\n %f  %f
        //     %f\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n\n\n", _dp.x, _dp.y, _dp.z, _dt0.x,
        //     _dt0.y, _dt0.z,
        //         _dt1.x, _dt1.y, _dt1.z, _dt2.x, _dt2.y, _dt2.z, _p.x, _p.y, _p.z,
        //         _t0.x, _t0.y, _t0.z, _t1.x, _t1.y, _t1.z, _t2.x, _t2.y, _t2.z);
        Scalar toc_lower_bound = (1 - eta) * (dist2_cur - thickness * thickness) /
                                 ((dist_cur + thickness) * max_disp_mag);

        p = __MATHUTILS__::__add(p, __MATHUTILS__::__s_vec_multiply(dp, toc_lower_bound));
        t0 = __MATHUTILS__::__add(t0,
                                  __MATHUTILS__::__s_vec_multiply(dt0, toc_lower_bound));
        t1 = __MATHUTILS__::__add(t1,
                                  __MATHUTILS__::__s_vec_multiply(dt1, toc_lower_bound));
        t2 = __MATHUTILS__::__add(t2,
                                  __MATHUTILS__::__s_vec_multiply(dt2, toc_lower_bound));

        dist2_cur = point_triangle_distance_unclassified(p, t0, t1, t2);
        dist_cur = sqrt(dist2_cur);
        if (toc && ((dist2_cur - thickness * thickness) / (dist_cur + thickness) < gap)) {
            break;
        }

        toc += toc_lower_bound;
        if (toc > 1.0) {
            return 1.0;
        }
    }
    return toc;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

typedef struct {
    Scalar3 ad, bd, cd, pd;
    Scalar3 a0, b0, c0, p0;
} NewtonCheckData;

inline __device__ bool _insideTriangle(Scalar3 a, Scalar3 b, Scalar3 c, Scalar3 p) {
    Scalar3 n, da, db, dc;
    Scalar wa, wb, wc;

    Scalar3 ba = __MATHUTILS__::__minus(b, a);
    Scalar3 ca = __MATHUTILS__::__minus(c, a);

    n = __MATHUTILS__::__v_vec_cross(ba, ca);  // cross(ba, ca);

    da = __MATHUTILS__::__minus(a, p);
    db = __MATHUTILS__::__minus(b, p);
    dc = __MATHUTILS__::__minus(c, p);
    // da = a - p, db = b - p, dc = c - p;
    if ((wa = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__v_vec_cross(db, dc), n)) < 0.0f)
        return false;
    if ((wb = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__v_vec_cross(dc, da), n)) < 0.0f)
        return false;
    if ((wc = __MATHUTILS__::__v_vec_dot(__MATHUTILS__::__v_vec_cross(da, db), n)) < 0.0f)
        return false;

    // Compute barycentric coordinates
    Scalar area2 = __MATHUTILS__::__v_vec_dot(n, n);
    wa /= area2, wb /= area2, wc /= area2;

    return true;
}

inline __device__ int solveQuadric(Scalar c[3], Scalar s[2], const Scalar& errorRate) {
    Scalar p, q, D;

    // make sure we have a d2 equation

    if (c[2] < errorRate) {
        if ((c[1]) < errorRate) return 0;
        s[0] = -c[0] / c[1];
        return 1;
    }

    // normal for: x^2 + px + q
    p = c[1] / (2.0f * c[2]);
    q = c[0] / c[2];
    D = p * p - q;

    if ((D) < errorRate) {
        // one float root
        s[0] = s[1] = -p;
        return 1;
    }

    if (D < 0.0f)
        // no real root
        return 0;

    else {
        // two real roots
        Scalar sqrt_D = sqrt(D);
        s[0] = sqrt_D - p;
        s[1] = -sqrt_D - p;
        return 2;
    }
}
inline __device__ int solveCubic(Scalar c[4], Scalar s[3], const Scalar& errorRate) {
    int i, num;
    Scalar sub, A, B, C, sq_A, p, q, cb_p, D;

    if (c[3] < errorRate) {
        return solveQuadric(c, s, errorRate);
    }

    A = c[2] / c[3];
    B = c[1] / c[3];
    C = c[0] / c[3];

    sq_A = A * A;
    Scalar ONE_DIV_3 = 1.0 / 3;
    p = ONE_DIV_3 * (-ONE_DIV_3 * sq_A + B);
    q = 0.5f * (2.0f / 27.0f * A * sq_A - ONE_DIV_3 * A * B + C);

    // use Cardano's formula

    cb_p = p * p * p;
    D = q * q + cb_p;
    if ((D) < errorRate) {
        if ((q) < errorRate) {
            // one triple solution
            s[0] = 0.0f;
            num = 1;
        } else {
            // one single and one float solution
            Scalar u = cbrt(-q);
            s[0] = 2.0f * u;
            s[1] = -u;
            num = 2;
        }
    } else if (D < 0.0f) {
        // casus irreductibilis: three real solutions
        Scalar phi = ONE_DIV_3 * acos(-q / sqrt(-cb_p));
        Scalar t = 2.0f * sqrt(-p);
        s[0] = t * cos(phi);
        s[1] = -t * cos(phi + __MATHUTILS__::__PI() / 3.0f);
        s[2] = -t * cos(phi - __MATHUTILS__::__PI() / 3.0f);
        num = 3;
    } else {
        // one real solution
        Scalar sqrt_D = sqrt(D);
        Scalar u = cbrt(sqrt_D + fabs(q));
        if (q > 0.0f)
            s[0] = -u + p / u;
        else
            s[0] = u - p / u;
        num = 1;
    }

    // resubstitute
    sub = ONE_DIV_3 * A;
    for (i = 0; i < num; i++) s[i] -= sub;
    return num;
}

inline __device__ void _equateCubic_VF(Scalar3 a0, Scalar3 ad, Scalar3 b0, Scalar3 bd,
                                       Scalar3 c0, Scalar3 cd, Scalar3 p0, Scalar3 pd,
                                       Scalar& a, Scalar& b, Scalar& c, Scalar& d,
                                       const Scalar& thickness) {
    Scalar3 dab, dac, dap;
    Scalar3 oab, oac, oap;
    Scalar3 dabXdac, dabXoac, oabXdac, oabXoac;

    dab = __MATHUTILS__::__minus(bd, ad), dac = __MATHUTILS__::__minus(cd, ad),
    dap = __MATHUTILS__::__minus(pd, ad);
    oab = __MATHUTILS__::__minus(b0, a0), oac = __MATHUTILS__::__minus(c0, a0),
    oap = __MATHUTILS__::__minus(p0, a0);

    dabXdac = __MATHUTILS__::__v_vec_cross(dab, dac);
    dabXoac = __MATHUTILS__::__v_vec_cross(dab, oac);
    oabXdac = __MATHUTILS__::__v_vec_cross(oab, dac);
    oabXoac = __MATHUTILS__::__v_vec_cross(oab, oac);

    a = __MATHUTILS__::__v_vec_dot(dap, dabXdac);
    b = __MATHUTILS__::__v_vec_dot(oap, dabXdac) +
        __MATHUTILS__::__v_vec_dot(dap, __MATHUTILS__::__add(dabXoac, oabXdac));
    c = __MATHUTILS__::__v_vec_dot(dap, oabXoac) +
        __MATHUTILS__::__v_vec_dot(oap, __MATHUTILS__::__add(dabXoac, oabXdac));
    d = thickness * __MATHUTILS__::__v_vec_dot(oap, oabXoac);
}

inline __device__ Scalar IntersectVF(Scalar3 ta0, Scalar3 tb0, Scalar3 tc0, Scalar3 ad,
                                     Scalar3 bd, Scalar3 cd, Scalar3 q0, Scalar3 qd,
                                     const Scalar& errorRate, const Scalar& thickness) {
    Scalar collisionTime = 1.0;

    Scalar a, b, c, d; /* cubic polynomial coefficients */
    _equateCubic_VF(ta0, ad, tb0, bd, tc0, cd, q0, qd, a, b, c, d, thickness);

    // if ((a) < errorRate && (b) < errorRate && (c) < errorRate && (d) < errorRate)
    //     return 1.0;

    Scalar roots[3];
    Scalar coeffs[4];
    coeffs[3] = a, coeffs[2] = b, coeffs[1] = c, coeffs[0] = d;
    int num;  // = solveCubic(coeffs, roots, errorRate);

    __MATHUTILS__::__NewtonSolverForCubicEquation(a, b, c, d, roots, num, errorRate);

    if (num == 0) return 1.0;

    for (int i = 0; i < num; i++) {
        Scalar r = roots[i];
        if (r < 0 || r > 1) continue;

        if (_insideTriangle(
                __MATHUTILS__::__add(__MATHUTILS__::__s_vec_multiply(ad, r), ta0),
                __MATHUTILS__::__add(__MATHUTILS__::__s_vec_multiply(bd, r), tb0),
                __MATHUTILS__::__add(__MATHUTILS__::__s_vec_multiply(cd, r), tc0),
                __MATHUTILS__::__add(__MATHUTILS__::__s_vec_multiply(qd, r), q0))) {
            if (collisionTime > r) {
                collisionTime = r;
            }
        }
    }

    return collisionTime;
}

__device__ Scalar doCCDVF(const Scalar3& _p, const Scalar3& _t0, const Scalar3& _t1,
                          const Scalar3& _t2, const Scalar3& _dp, const Scalar3& _dt0,
                          const Scalar3& _dt1, const Scalar3& _dt2, Scalar errorRate,
                          Scalar thickness) {
    Scalar ret =
        IntersectVF(_t0, _t1, _t2, _dt0, _dt1, _dt2, _p, _dp, errorRate, thickness);

    return ret;
}

}; // namespace ACCD