#include "hip/hip_runtime.h"
//
// device_fem_data.cu
// GIPC
//
// created by Kemeng Huang on 2022/12/01
// Copyright (c) 2024 Kemeng Huang. All rights reserved.
//

#include "device_fem_data.cuh"
#include "cuda_tools.h"


void device_TetraData::Malloc_DEVICE_MEM(const int& vertex_num, const int& tetradedra_num, const int& triangle_num, const int& softNum, const int& tri_edgeNum){
	int maxNumbers = vertex_num > tetradedra_num ? vertex_num : tetradedra_num;
	CUDA_SAFE_CALL(hipMalloc((void**)&vertexes, vertex_num*sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&o_vertexes, vertex_num * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&velocities, vertex_num * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&rest_vertexes, vertex_num * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&temp_double3Mem, vertex_num * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&xTilta, vertex_num * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fb, vertex_num * sizeof(double3)));
	
	CUDA_SAFE_CALL(hipMalloc((void**)&tetrahedras, tetradedra_num * sizeof(uint4)));
	CUDA_SAFE_CALL(hipMalloc((void**)&tempTetrahedras, tetradedra_num * sizeof(uint4)));


	CUDA_SAFE_CALL(hipMalloc((void**)&tri_edges, tri_edgeNum * sizeof(uint2)));
	CUDA_SAFE_CALL(hipMalloc((void**)&tri_edge_adj_vertex, tri_edgeNum * sizeof(uint2)));


	CUDA_SAFE_CALL(hipMalloc((void**)&volum, tetradedra_num * sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc((void**)&masses, vertex_num * sizeof(double)));

	CUDA_SAFE_CALL(hipMalloc((void**)&tempDouble, maxNumbers * sizeof(double)));
	//CUDA_SAFE_CALL(hipMalloc((void**)&tempM, vertex_num * sizeof(double)));

	CUDA_SAFE_CALL(hipMalloc((void**)&MChash, maxNumbers * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)&sortIndex, maxNumbers * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)&BoundaryType, vertex_num * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&tempBoundaryType, vertex_num * sizeof(int)));

	CUDA_SAFE_CALL(hipMemset(BoundaryType, 0, vertex_num * sizeof(int)));

	//CUDA_SAFE_CALL(hipMalloc((void**)&sortVertIndex, vertex_num * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)&sortMapVertIndex, vertex_num * sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMalloc((void**)&DmInverses, tetradedra_num * sizeof(__GEIGEN__::Matrix3x3d)));
	CUDA_SAFE_CALL(hipMalloc((void**)&Constraints, vertex_num * sizeof(__GEIGEN__::Matrix3x3d)));

	CUDA_SAFE_CALL(hipMalloc((void**)&tempMat3x3, maxNumbers * sizeof(__GEIGEN__::Matrix3x3d)));
	//CUDA_SAFE_CALL(hipMalloc((void**)&tempConstraints, vertex_num * sizeof(__GEIGEN__::Matrix3x3d)));

	

	CUDA_SAFE_CALL(hipMalloc((void**)&targetIndex, softNum * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc((void**)&targetVert, softNum * sizeof(double3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&triDmInverses, triangle_num * sizeof(__GEIGEN__::Matrix2x2d)));
	CUDA_SAFE_CALL(hipMalloc((void**)&area, triangle_num * sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc((void**)&triangles, triangle_num * sizeof(uint4)));
}

device_TetraData::~device_TetraData() {
	FREE_DEVICE_MEM();
}

void device_TetraData::FREE_DEVICE_MEM() {
	CUDA_SAFE_CALL(hipFree(sortIndex));
	CUDA_SAFE_CALL(hipFree(sortMapVertIndex));
	CUDA_SAFE_CALL(hipFree(vertexes)); 
	CUDA_SAFE_CALL(hipFree(o_vertexes));
	CUDA_SAFE_CALL(hipFree(temp_double3Mem));
	CUDA_SAFE_CALL(hipFree(velocities));
	CUDA_SAFE_CALL(hipFree(rest_vertexes));
	CUDA_SAFE_CALL(hipFree(xTilta));
	CUDA_SAFE_CALL(hipFree(fb));
	CUDA_SAFE_CALL(hipFree(tetrahedras));
	CUDA_SAFE_CALL(hipFree(tempTetrahedras));
	CUDA_SAFE_CALL(hipFree(volum));
	CUDA_SAFE_CALL(hipFree(masses));
	CUDA_SAFE_CALL(hipFree(DmInverses));
	CUDA_SAFE_CALL(hipFree(Constraints));
	CUDA_SAFE_CALL(hipFree(tempMat3x3));
	CUDA_SAFE_CALL(hipFree(MChash));
	CUDA_SAFE_CALL(hipFree(tempDouble));
	CUDA_SAFE_CALL(hipFree(BoundaryType));
	CUDA_SAFE_CALL(hipFree(tempBoundaryType));

	CUDA_SAFE_CALL(hipFree(targetIndex));
	CUDA_SAFE_CALL(hipFree(targetVert));
	CUDA_SAFE_CALL(hipFree(triDmInverses));
	CUDA_SAFE_CALL(hipFree(area));
	CUDA_SAFE_CALL(hipFree(triangles));

	CUDA_SAFE_CALL(hipFree(tri_edges));
	CUDA_SAFE_CALL(hipFree(tri_edge_adj_vertex));
}


